#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include "h1.h"
#include <iomanip>
#include <cstdlib>
#include <libconfig.h++>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "cudaParticle.h"
#include "boris.h"
#include "ionize.h"
#include <algorithm>
#include <boost/timer/timer.hpp>

using namespace std;
using namespace libconfig;
using namespace boost::timer;

__host__ __device__
 cudaParticle       generateParticle(double x1){
		cudaParticle p; 
               p.x = x1;
               p.y = 2.0;
               p.z = 3.0;
		
		return p;
        };

int main()
{

Config cfg;

cfg.readFile("gitrInput.cfg");

char outname[] = "Deposition.m";
char outnameCharge[] = "Charge.m";
char outnameEnergy[] = "Energy.m";

// Volume definition

double xMinV = cfg.lookup("volumeDefinition.xMinV");
double xMaxV = cfg.lookup("volumeDefinition.xMaxV");
cout << "xMaxV  " << xMaxV << endl;
	// grid
int nXv = cfg.lookup("volumeDefinition.grid.nXv");
int nYv = cfg.lookup("volumeDefinition.grid.nYv");
int nZv = cfg.lookup("volumeDefinition.grid.nZv");

// Surface definition

double yMin = cfg.lookup("surfaceDefinition.yMin");
double yMax = cfg.lookup("surfaceDefinition.yMax");

double zMin = cfg.lookup("surfaceDefinition.zMin");
double zMax  = cfg.lookup("surfaceDefinition.zMax");



// Surface grid

int nY  = cfg.lookup("surfaceDefinition.grid.nY");
int nZ  = cfg.lookup("surfaceDefinition.grid.nZ");
// Surface parameterization z = dz/dx * x + b

double surface_dz_dx  = cfg.lookup("surfaceDefinition.planeParameterization.surface_dz_dx");
double surface_zIntercept = cfg.lookup("surfaceDefinition.planeParameterization.surface_zIntercept");

// Constant B field value - only used when BfieldInterpolator_number = 0
double Bx_in = cfg.lookup("bField.Bx_in");
double By_in = cfg.lookup("bField.By_in");
double Bz_in = cfg.lookup("bField.Bz_in");
double connectionLength = cfg.lookup("bField.connectionLength");

// Particle time stepping control

int ionization_nDtPerApply  = cfg.lookup("timeStep.ionization_nDtPerApply");
int collision_nDtPerApply  = cfg.lookup("timeStep.collision_nDtPerApply");
cout << "collision_nDtPerApply  " << collision_nDtPerApply << endl;
// Perp DiffusionCoeff - only used when Diffusion interpolator is = 0
double perDiffusionCoeff_in;

// Background profile values used Density, temperature interpolators are 0
// or 2
double densitySOLDecayLength;
double tempSOLDecayLength;

// Background species info
int *densityChargeBins;
int *background_Z;
double *background_amu;
double *background_flow;
double *maxDensity;
double *maxTemp_eV;

Setting& backgroundPlasma = cfg.lookup("backgroundPlasma");
int nS = backgroundPlasma["Z"].getLength();

cout << "nS  " << nS << endl;

Setting& diagnostics = cfg.lookup("diagnostics");
int nDensityChargeBins = diagnostics["densityChargeBins"].getLength();

cout << "nDensityChargeBins  " << nDensityChargeBins << endl;

densityChargeBins = new int[nDensityChargeBins];

background_Z = new int[nS];
background_amu = new double[nS];
background_flow = new double[nS];
maxDensity = new double[nS];
maxTemp_eV = new double[nS];

for(int i=0; i<nS; i++)
{
background_Z[i] = backgroundPlasma["Z"][i];
background_amu[i] = backgroundPlasma["amu"][i];
background_flow[i] = backgroundPlasma["flow"]["fractionOfThermalVelocity"][i];
maxDensity[i] = backgroundPlasma["density"]["max"][i];
maxTemp_eV[i] = backgroundPlasma["temp"]["max"][i];

cout << maxTemp_eV[i];
 }

double x = cfg.lookup("impurityParticleSource.initialConditions.x_start");
double y = cfg.lookup("impurityParticleSource.initialConditions.y_start");
double z = cfg.lookup("impurityParticleSource.initialConditions.z_start");

double Ex = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_x_start");
double Ey = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_y_start");
double Ez = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_z_start");

double amu = cfg.lookup("impurityParticleSource.initialConditions.impurity_amu");
double Z = cfg.lookup("impurityParticleSource.initialConditions.impurity_Z");

	double **SurfaceBins;
	double **SurfaceBinsCharge;
	double **SurfaceBinsEnergy;
	
	SurfaceBins = new double*[nY];
	SurfaceBinsCharge = new double*[nY];
	SurfaceBinsEnergy = new double*[nY];

 	SurfaceBins[0] = new double[nY*nZ];
 	SurfaceBinsCharge[0] = new double[nY*nZ];
 	SurfaceBinsEnergy[0] = new double[nY*nZ];
			
	for(int i=0 ; i<nY ; i++)
				{
					SurfaceBins[i] = &SurfaceBins[0][i*nZ];
					SurfaceBinsCharge[i] = &SurfaceBinsCharge[0][i*nZ];
					SurfaceBinsEnergy[i] = &SurfaceBinsEnergy[0][i*nZ];
					
				for(int j=0 ; j<nZ ; j++)
					{
						SurfaceBins[i][j] = 0;
						SurfaceBinsCharge[i][j] = 0;
						SurfaceBinsEnergy[i][j] = 0;
					}
				}
	
	//double dt;
	//double nPtsPerGyroOrbit = cfg.lookup("timeStep.nPtsPerGyroOrbit");
	//dt = 1e-6/nPtsPerGyroOrbit;

	int nP = cfg.lookup("impurityParticleSource.nP");
 	cout << "Number of particles: " << nP << endl;				
	Particle Particles[nP];
	INIT(nP,Particles, cfg);

	unsigned long seed=(unsigned long)(time(NULL));
	srand(seed);
	
	int nT = cfg.lookup("timeStep.nT");
    cout << "Number of time steps: " << nT << endl;	
    
    int surfaceIndexY;
	int surfaceIndexZ;

	thrust::host_vector<int> H(4); 
	H[0] = 14;
 	H[1] = 20;
	H[2] = 38;
	H[3] = 46;

	thrust::device_vector<int> D = H;
	D[0] = 99;
	D[1] = 88;  

	for(int i = 0; i < D.size(); i++)
	    std::cout << "D[" << i << "] = " << D[i] << std::endl;

    float dt = 1e-6;
	cudaParticle p1(x,y,z,Ex,Ey,Ez,Z,amu,dt);

    long nParticles = 1e5;
    std::cout << "nParticles: " << nParticles << std::endl;
	thrust::host_vector<cudaParticle> hostCudaParticleVector(nParticles,p1);

	//for(int i=0; i < hostCudaParticleVector.size(); i++)
	//    std::cout << hostCudaParticleVector[i].x << std::endl;

    cpu_timer timer;

	std::cout << "Initial x position GPU: " << hostCudaParticleVector[1].x << std::endl;

	thrust::device_vector<cudaParticle> deviceCudaParticleVector = hostCudaParticleVector;

    hipDeviceSynchronize();

    cpu_times copyToDeviceTime = timer.elapsed();
    std::cout << "copyToDeviceTime: " << copyToDeviceTime.wall*1e-9 << '\n';

    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), move_boris() );

    hipDeviceSynchronize();

    cpu_times moveTimeGPU = timer.elapsed();
    std::cout << "moveTimeGPU: " << (moveTimeGPU.wall-copyToDeviceTime.wall)*1e-9 << '\n';

    //thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), ionize() );

    //cpu_times ionizeTimeGPU = timer.elapsed();
    //std::cout << "ionizeTimeGPU: " << ionizeTimeGPU.wall*1e-9 << '\n';

    thrust::host_vector<cudaParticle> hostCudaParticleVector2 = deviceCudaParticleVector;

    hipDeviceSynchronize();

    cpu_times copyToHostTime = timer.elapsed();
    std::cout << "copyToHostTime: " << (copyToHostTime.wall-moveTimeGPU.wall)*1e-9 << '\n';
	std::cout << "Final x position GPU: " << hostCudaParticleVector2.back().x << std::endl;

    // CPU

	std::vector<cudaParticle> particleVector(nParticles,p1);

    cpu_times createParticlesTimeCPU = timer.elapsed();
    std::cout << "createParticesTimeCPU: " << (createParticlesTimeCPU.wall-copyToHostTime.wall)*1e-9 << '\n';

    std::for_each( particleVector.begin(), particleVector.end(), move_boris() );

    cpu_times moveTimeCPU = timer.elapsed();
    std::cout << "moveTimeCPU: " << (moveTimeCPU.wall-createParticlesTimeCPU.wall)*1e-9 << '\n';

    std::cout << "GPU Speedup: " << (moveTimeCPU.wall-createParticlesTimeCPU.wall) / (moveTimeGPU.wall-copyToDeviceTime.wall) << '\n';
	std::cout << "Final x position CPU: " << particleVector.back().x << std::endl;
	return 0;
}
