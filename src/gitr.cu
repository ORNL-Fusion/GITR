#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include "h1.cuh"
#include <iomanip>
#include <cstdlib>
#include <libconfig.h++>
#include "boris.h"
#include "geometryCheck.h"
#include "ionize.h"
#include "recombine.h"
#include "crossFieldDiffusion.h"
#include "coulombCollisions.h"
#include "thermalForce.h"
#include "surfaceModel.h"
#include "interp2d.hpp"
#include <algorithm>
#include <random>
#include "Particle.h"
#include "Boundary.h"
#include <boost/timer/timer.hpp>
#include <vector>
#include "io.hpp"
#include "testRoutine.h"
#include "testRoutineCuda.h"
#include "boundaryInit.h"

#ifdef __HIPCC__
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#endif

using namespace std;
using namespace libconfig;
using namespace boost::timer;

int main()
{

Config cfg,cfg_geom;

cfg.readFile("gitrInput.cfg");
cfg_geom.readFile("gitrGeometry.cfg");

int nBx = 200;
int nBy = 200;
    double **Bfield;
    
    Bfield = new double*[nBx];

    Bfield[0] = new double[nBx*nBy];
            
    for(int i=0 ; i<nBx ; i++)
    {
        Bfield[i] = &Bfield[0][i*nBy];
        for(int j=0 ; j<nBy ; j++)
        {
            Bfield[i][j] = 0;
        }
    }
string fileName("ar2Input.nc");
int a = read_ar2Input(fileName, Bfield);
std::cout << "Bfield array " <<Bfield[100][100] << Bfield[134][76]<<Bfield[76][134] << std::endl;

string BprofileName("ar2Input.nc");
string bfieldNrString("nR");
string bfieldNzString("nZ");
string bfieldRString("br");
string bfieldGridrName("r");
string bfieldGridzName("z");
int nR;
int nZ;
int b1 = read_profileNs(BprofileName,bfieldNrString,bfieldNzString,nR,nZ);
std::vector<double> bfieldGridr(nR), bfieldGridz(nZ);
std::vector<double> br(nR*nZ);
int b2 = read_profiles(BprofileName,nR,nZ,bfieldGridrName, bfieldGridr,bfieldGridzName, bfieldGridz,bfieldRString, br);
string bfieldZString("bz");
std::vector<double> bz(nR*nZ);
int b3 = read_profiles(BprofileName,nR,nZ,bfieldGridrName, bfieldGridr,bfieldGridzName, bfieldGridz,bfieldZString, bz);
string bfieldTString("bt");
std::vector<double> bt(nR*nZ);
int b4 = read_profiles(BprofileName,nR,nZ,bfieldGridrName, bfieldGridr,bfieldGridzName, bfieldGridz,bfieldTString, bt);

string profileName("profiles.nc");
string densNxString("n_x");
string densNzString("n_z");
string densString("ne");
string densGridxName("gridx");
string densGridzName("gridz");
int n_x;
int n_z;
int a1 = read_profileNs(profileName,densNxString,densNzString,n_x,n_z);
std::vector<double> gridx(n_x), gridz(n_z);
std::vector<double> dens(n_x*n_z);
int a2 = read_profiles(profileName,n_x,n_z,densGridxName, gridx,densGridzName, gridz,densString, dens);



thrust::device_vector<double> device_dens = dens;
thrust::device_vector<double> device_gridx = gridx;
thrust::device_vector<double> device_gridz = gridz;
double interp_val1 = interp2d(2.25, 0.0, -1.3, &gridx, &gridz, &dens);
std::cout << "interpolated value " << interp_val1 << std::endl;
std::vector<double> doubleVector(5,1.1);
thrust::device_vector<double> dd = doubleVector;

std::cout << "starting print loop" << std::endl;
std::for_each(doubleVector.begin(), doubleVector.end(), test_routine(2.25, 0.0, -1.3, &gridx, &gridz, &dens) );
for (int i=0; i<5; i++)
{
        std::cout << "gridx: " << gridx[i] << std::endl;
}

double* gxptr2 = thrust::raw_pointer_cast(device_gridx.data());
double* gzptr2 = thrust::raw_pointer_cast(device_gridz.data());
double* dtptr2 = thrust::raw_pointer_cast(device_dens.data());
thrust::for_each(dd.begin(), dd.end(), test_routinecuda(2.25, 0.0, -1.3,n_x,n_z, gxptr2, gzptr2, dtptr2) );
thrust::host_vector<double> doubleVector2 = dd;
#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif
for (int i=0; i<5; i++)
{
        std::cout << "device doubleVector values: " << doubleVector2[i] << std::endl;
}


int nCS = 74;
int nTemperaturesIonize = 24;
int nDensitiesIonize = 24;
std::vector<double> coeffArray(nCS*nTemperaturesIonize*nDensitiesIonize);
string ADASName("ADAS_Rates_W.nc");
string IonizCoeffString("IonizationRateCoeff");
string gridTionizeName("gridTemperature_Ionization");
string gridNionizeName("gridDensity_Ionization");
std::vector<double> gridTemperature_Ionization(nTemperaturesIonize), gridDensity_Ionization(nDensitiesIonize);

int    a3 = read_profiles(ADASName, nTemperaturesIonize,nDensitiesIonize,gridTionizeName, 
        gridTemperature_Ionization,gridNionizeName,
        gridDensity_Ionization,
        IonizCoeffString,
        coeffArray);
        std::cout << "Coeff vector print " << 
        coeffArray[0*nTemperaturesIonize*nDensitiesIonize+ 1*nTemperaturesIonize+ 0] << std::endl;




char outname[] = "Deposition.m";
char outnameCharge[] = "Charge.m";
char outnameEnergy[] = "Energy.m";

//Geometry Definition
Setting& geom = cfg_geom.lookup("geom");
int nLines = geom["x1"].getLength();
std::cout << "Number of Geometric Objects Loaded: " << nLines << std::endl;
#ifdef __HIPCC__
        thrust::host_vector<Boundary> hostBoundaryVector(nLines+1);
#else
        std::vector<Boundary> hostBoundaryVector(nLines+1);
#endif
for(int i=0 ; i<nLines ; i++)
    {
     hostBoundaryVector[i].x1 = geom["x1"][i];
     hostBoundaryVector[i].z1 = geom["z1"][i];
     hostBoundaryVector[i].x2 = geom["x2"][i];
     hostBoundaryVector[i].z2 = geom["z2"][i];
     hostBoundaryVector[i].Z = geom["Z"][i];
     hostBoundaryVector[i].slope_dzdx = geom["slope"][i];
     hostBoundaryVector[i].intercept_z = geom["intercept"][i];
     hostBoundaryVector[i].length = geom["length"][i];
    }   
hostBoundaryVector[nLines].Z = geom["Z"][nLines];
hostBoundaryVector[nLines].y1 = geom["y1"];
hostBoundaryVector[nLines].y2 = geom["y2"];
hostBoundaryVector[nLines].periodic = geom["periodic"];
std::for_each(hostBoundaryVector.begin(), hostBoundaryVector.end()-1, boundary_init(&gridx,&gridz,&dens,&bfieldGridr,&bfieldGridz,&br,&bz, &bt) );
std::cout << "exited bound_init" << std::endl;

#ifdef __HIPCC__
    thrust::device_vector<Boundary> deviceBoundaryVector = hostBoundaryVector;
    Boundary * BoundaryDevicePointer = thrust::raw_pointer_cast(deviceBoundaryVector.data());
#else
    std::vector<Boundary> * BoundaryHostPointer = &hostBoundaryVector;    
#endif
    // Volume definition

double xMinV = cfg.lookup("volumeDefinition.xMinV");
double xMaxV = cfg.lookup("volumeDefinition.xMaxV");
    // grid
int nXv = cfg.lookup("volumeDefinition.grid.nXv");
int nYv = cfg.lookup("volumeDefinition.grid.nYv");
int nZv = cfg.lookup("volumeDefinition.grid.nZv");

// Surface definition

double yMin = cfg.lookup("surfaceDefinition.yMin");
double yMax = cfg.lookup("surfaceDefinition.yMax");

double zMin = cfg.lookup("surfaceDefinition.zMin");
double zMax  = cfg.lookup("surfaceDefinition.zMax");


// Surface parameterization z = dz/dx * x + b

double surface_dz_dx  = cfg.lookup("surfaceDefinition.planeParameterization.surface_dz_dx");
double surface_zIntercept = cfg.lookup("surfaceDefinition.planeParameterization.surface_zIntercept");

// Constant B field value - only used when BfieldInterpolator_number = 0
double Bx_in = cfg.lookup("bField.Bx_in");
double By_in = cfg.lookup("bField.By_in");
double Bz_in = cfg.lookup("bField.Bz_in");
double connectionLength = cfg.lookup("bField.connectionLength");

// Particle time stepping control

int ionization_nDtPerApply  = cfg.lookup("timeStep.ionization_nDtPerApply");
int collision_nDtPerApply  = cfg.lookup("timeStep.collision_nDtPerApply");
// Perp DiffusionCoeff - only used when Diffusion interpolator is = 0
double perDiffusionCoeff_in = cfg.lookup("perpDiffusion.perDiffusionCoeff_in");

// Background species info
int *background_Z;
double *background_amu;
double *background_flow;
double *maxDensity;
double *maxTemp_eV;

#ifdef __HIPCC__
    cout<<"Using THRUST"<<endl;
#else
    cout<<"Not using THRUST"<<endl;
#endif

Setting& backgroundPlasma = cfg.lookup("backgroundPlasma");
int nS = backgroundPlasma["Z"].getLength();

Setting& diagnostics = cfg.lookup("diagnostics");


background_Z = new int[nS];
background_amu = new double[nS];
background_flow = new double[nS];
maxDensity = new double[nS];
maxTemp_eV = new double[nS];

for(int i=0; i<nS; i++)
{
background_Z[i] = backgroundPlasma["Z"][i];
background_amu[i] = backgroundPlasma["amu"][i];
background_flow[i] = backgroundPlasma["flow"]["fractionOfThermalVelocity"][i];
maxDensity[i] = backgroundPlasma["density"]["max"][i];
maxTemp_eV[i] = backgroundPlasma["temp"]["max"][i];
}

    double x = cfg.lookup("impurityParticleSource.initialConditions.x_start");
    double y = cfg.lookup("impurityParticleSource.initialConditions.y_start");
    double z = cfg.lookup("impurityParticleSource.initialConditions.z_start");
    
    double Ex = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_x_start");
    double Ey = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_y_start");
    double Ez = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_z_start");
    
    double amu = cfg.lookup("impurityParticleSource.initialConditions.impurity_amu");
    double Z = cfg.lookup("impurityParticleSource.initialConditions.impurity_Z");
/*
    double **SurfaceBins;
    double **SurfaceBinsCharge;
    double **SurfaceBinsEnergy;
    double **SurfaceBinsErosion;
    
    SurfaceBins = new double*[nY];
    SurfaceBinsCharge = new double*[nY];
    SurfaceBinsEnergy = new double*[nY];
    SurfaceBinsErosion = new double*[nY];

    SurfaceBins[0] = new double[nY*nZ];
    SurfaceBinsCharge[0] = new double[nY*nZ];
    SurfaceBinsEnergy[0] = new double[nY*nZ];
    SurfaceBinsErosion[0] = new double[nY*nZ];
            
    for(int i=0 ; i<nY ; i++)
    {
        SurfaceBins[i] = &SurfaceBins[0][i*nZ];
        SurfaceBinsCharge[i] = &SurfaceBinsCharge[0][i*nZ];
        SurfaceBinsEnergy[i] = &SurfaceBinsEnergy[0][i*nZ];
        SurfaceBinsErosion[i] = &SurfaceBinsErosion[0][i*nZ];               
        for(int j=0 ; j<nZ ; j++)
        {
            SurfaceBins[i][j] = 0;
            SurfaceBinsCharge[i][j] = 0;
            SurfaceBinsEnergy[i][j] = 0;
            SurfaceBinsErosion[i][j] = 0;
        }
    }
*/    
    double dt;
    double nPtsPerGyroOrbit = cfg.lookup("timeStep.nPtsPerGyroOrbit");
    dt = 1e-6/nPtsPerGyroOrbit;

    int nP = cfg.lookup("impurityParticleSource.nP");
    cout << "Number of particles: " << nP << endl;              
    long nParticles = nP;
    int nT = cfg.lookup("timeStep.nT");
    cout << "Number of time steps: " << nT << " With dt = " << dt << endl; 
    
//    int surfaceIndexY;
//    int surfaceIndexZ;
#if PARTICLE_SOURCE == 0
    Particle p1(x,y,z,Ex,Ey,Ez,Z,amu);
#endif

#ifdef __HIPCC__
      thrust::host_vector<Particle> hostCudaParticleVector(nParticles,p1);
#else
        std::vector<Particle> hostCudaParticleVector(nParticles,p1);
#endif

#if GEOM_TRACE > 0       
            std::uniform_real_distribution<float> dist2(0,1);
            std::random_device rd2;
            std::cout << "Randomizing velocities to trace geometry. " << std::endl;
       
      for (int i=0 ; i<nParticles ; i++)
            {   double theta = dist2(rd2)*2*3.1415;
                double phi = dist2(rd2)*3.1415;
                double mag = 2e3;
                hostCudaParticleVector[i].vx = mag*cos(theta)*sin(phi);
                hostCudaParticleVector[i].vy = mag*sin(theta)*sin(phi);
                hostCudaParticleVector[i].vz = mag*cos(phi);
            }
#endif
       
            cpu_timer timer;

#ifdef __HIPCC__
    thrust::device_vector<Particle> deviceCudaParticleVector = hostCudaParticleVector;
#endif

    std::uniform_real_distribution<float> dist(0,1e6);
        std::random_device rd;
        std::default_random_engine generator(rd());
    
#if USEIONIZATION > 0
    std::vector<float> seeds0(nP);
    std::generate( seeds0.begin(), seeds0.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
    thrust::device_vector<float> deviceSeeds0 = seeds0;
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds0.begin(), deviceCudaParticleVector.begin(), randInit(0) );
#else
    std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds0.begin(), hostCudaParticleVector.begin(), randInit(0) );
#endif
#endif

#if USERECOMBINATION > 0
        std::vector<float> seeds1(nP);
        std::generate( seeds1.begin(), seeds1.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds1 = seeds1;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds1.begin(), deviceCudaParticleVector.begin(), randInit(1) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds1.begin(), hostCudaParticleVector.begin(), randInit(1) );
#endif
#endif

#if USEPERPDIFFUSION > 0
        std::vector<float> seeds2(nP);
        std::generate( seeds2.begin(), seeds2.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds2 = seeds2;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds2.begin(), deviceCudaParticleVector.begin(), randInit(2) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds2.begin(), hostCudaParticleVector.begin(), randInit(2) );
#endif
#endif

#if USECOULOMBCOLLISIONS > 0
        std::vector<float> seeds3(nP),seeds4(nP),seeds5(nP);
        std::generate( seeds3.begin(), seeds3.end(), [&]() { return dist(generator); } );
    std::generate( seeds4.begin(), seeds4.end(), [&]() { return dist(generator); } );
    std::generate( seeds5.begin(), seeds5.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds3 = seeds3,deviceSeeds4 = seeds4,deviceSeeds5 = seeds5;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds3.begin(), deviceCudaParticleVector.begin(), randInit(3) );
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds4.begin(), deviceCudaParticleVector.begin(), randInit(4) );
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds5.begin(), deviceCudaParticleVector.begin(), randInit(5) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds3.begin(), hostCudaParticleVector.begin(), randInit(3) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds4.begin(), hostCudaParticleVector.begin(), randInit(4) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds5.begin(), hostCudaParticleVector.begin(), randInit(5) );
#endif
#endif

#if USESURFACEMODEL > 0
        std::vector<float> seeds6(nP);
        std::generate( seeds6.begin(), seeds6.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds6 = seeds6;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds6.begin(), deviceCudaParticleVector.begin(), randInit(6) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds6.begin(), hostCudaParticleVector.begin(), randInit(6) );
#endif
#endif
    
    cpu_times copyToDeviceTime = timer.elapsed();
    std::cout << "Initialize rand state and copyToDeviceTime: " << copyToDeviceTime.wall*1e-9 << '\n';
    for(int tt=0; tt< nT; tt++)
    {
#ifdef __HIPCC__
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), move_boris(dt,BoundaryDevicePointer, nLines) );
        try {
            thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), geometry_check(nLines,BoundaryDevicePointer) );
        }
        catch (thrust::system_error &e) {
            std::cerr << "Thrust system error: " << e.what() << std::endl;
            exit(-1);
        }
#if USEIONIZATION > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), ionize(dt) );
#endif
#if USERECOMBINATION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), crossFieldDiffusion(dt,perDiffusionCoeff_in));
#endif
#if USECOULOMBCOLLISIONS > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), coulombCollisions(dt) );
#endif
#if USETHERMALFORCE > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), thermalForce(dt) );
#endif
#else
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), move_boris(dt,hostBoundaryVector,nLines) );
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), geometry_check(nLines,hostBoundaryVector) );
#if USEIONIZATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), ionize(dt) );
#endif
#if USERECOMBINATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), crossFieldDiffusion(dt,perDiffusionCoeff_in));
#endif
#if USECOULOMBCOLLISIONS > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), coulombCollisions(dt) );
#endif
#if USETHERMALFORCE > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), thermalForce(dt) );
#endif
#endif
    }
    cpu_times ionizeTimeGPU = timer.elapsed();
    std::cout << "Particle Moving Time: " << ionizeTimeGPU.wall*1e-9 << '\n';

#ifdef __HIPCC__
    hostCudaParticleVector = deviceCudaParticleVector;
#endif

    for(int i=0; i < hostCudaParticleVector.size(); i++){
       //std::cout << " final pos" <<  i << " " <<hostCudaParticleVector[i].x << " " << hostCudaParticleVector[i].y << " " << hostCudaParticleVector[i].z << std::endl;
        /*if(hostCudaParticleVector[i].hitWall == 1){
        surfaceIndexY = int(floor((hostCudaParticleVector[i].y - yMin)/(yMax - yMin)*(nY) + 0.0f));
        surfaceIndexZ = int(floor((hostCudaParticleVector[i].z - zMin)/(zMax - zMin)*(nZ) + 0.0f));
        SurfaceBins[surfaceIndexY][surfaceIndexZ] +=  1.0 ;

        SurfaceBinsCharge[surfaceIndexY][surfaceIndexZ] += hostCudaParticleVector[i].Z ;
        SurfaceBinsEnergy[surfaceIndexY][surfaceIndexZ] += 0.5*hostCudaParticleVector[i].amu*1.6737236e-27*(hostCudaParticleVector[i].vx*hostCudaParticleVector[i].vx +  hostCudaParticleVector[i].vy*hostCudaParticleVector[i].vy+ hostCudaParticleVector[i].vz*hostCudaParticleVector[i].vz)/1.60217662e-19;
        } */ 
    }

//    OUTPUT( outname,nY, nZ, SurfaceBins);
//    OUTPUT( outnameCharge,nY, nZ, SurfaceBinsCharge);
//    OUTPUT( outnameEnergy,nY, nZ, SurfaceBinsEnergy);

    ofstream outfile2;
    outfile2.open ("positions.m");
    for(int i=1 ; i<=nP ; i++)
      {
        outfile2 << "Pos( " << i<< ",:) = [ " ;
        outfile2 << hostCudaParticleVector[i-1].x << " " << hostCudaParticleVector[i-1].y << " " << hostCudaParticleVector[i-1].z << " ];" << std::endl;
      }
       outfile2.close();


#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif

    cpu_times copyToHostTime = timer.elapsed();

    cpu_times createParticlesTimeCPU = timer.elapsed();
    std::cout << "Copy to host, bin and output time: " << (createParticlesTimeCPU.wall-copyToHostTime.wall)*1e-9 << '\n';
    return 0;
}
