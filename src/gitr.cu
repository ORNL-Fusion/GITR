#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include "h1.cuh"
#include <iomanip>
#include <cstdlib>
#include <libconfig.h++>
#include "boris.h"
#include "geometryCheck.h"
#include "ionize.h"
#include "recombine.h"
#include "crossFieldDiffusion.h"
#include "coulombCollisions.h"
#include "thermalForce.h"
#include "surfaceModel.h"
#include <algorithm>
#include <random>
#include "Particle.h"
#include <boost/timer/timer.hpp>

#ifdef __HIPCC__
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#endif

using namespace std;
using namespace libconfig;
using namespace boost::timer;

int main()
{

Config cfg;

cfg.readFile("gitrInput.cfg");

char outname[] = "Deposition.m";
char outnameCharge[] = "Charge.m";
char outnameEnergy[] = "Energy.m";

// Volume definition

double xMinV = cfg.lookup("volumeDefinition.xMinV");
double xMaxV = cfg.lookup("volumeDefinition.xMaxV");
cout << "xMaxV  " << xMaxV << endl;
    // grid
int nXv = cfg.lookup("volumeDefinition.grid.nXv");
int nYv = cfg.lookup("volumeDefinition.grid.nYv");
int nZv = cfg.lookup("volumeDefinition.grid.nZv");

// Surface definition

double yMin = cfg.lookup("surfaceDefinition.yMin");
double yMax = cfg.lookup("surfaceDefinition.yMax");

double zMin = cfg.lookup("surfaceDefinition.zMin");
double zMax  = cfg.lookup("surfaceDefinition.zMax");

// Surface grid

int nY  = cfg.lookup("surfaceDefinition.grid.nY");
int nZ  = cfg.lookup("surfaceDefinition.grid.nZ");
// Surface parameterization z = dz/dx * x + b

double surface_dz_dx  = cfg.lookup("surfaceDefinition.planeParameterization.surface_dz_dx");
double surface_zIntercept = cfg.lookup("surfaceDefinition.planeParameterization.surface_zIntercept");

// Constant B field value - only used when BfieldInterpolator_number = 0
double Bx_in = cfg.lookup("bField.Bx_in");
double By_in = cfg.lookup("bField.By_in");
double Bz_in = cfg.lookup("bField.Bz_in");
double connectionLength = cfg.lookup("bField.connectionLength");

// Particle time stepping control

int ionization_nDtPerApply  = cfg.lookup("timeStep.ionization_nDtPerApply");
int collision_nDtPerApply  = cfg.lookup("timeStep.collision_nDtPerApply");
cout << "collision_nDtPerApply  " << collision_nDtPerApply << endl;
// Perp DiffusionCoeff - only used when Diffusion interpolator is = 0
double perDiffusionCoeff_in = cfg.lookup("perpDiffusion.perDiffusionCoeff_in");

// Background profile values used Density, temperature interpolators are 0 or 2
double densitySOLDecayLength;
double tempSOLDecayLength;

// Background species info
int *densityChargeBins;
int *background_Z;
double *background_amu;
double *background_flow;
double *maxDensity;
double *maxTemp_eV;

#ifdef __HIPCC__
    cout<<"Using THRUST"<<endl;
#else
    cout<<"Not using THRUST"<<endl;
#endif

Setting& backgroundPlasma = cfg.lookup("backgroundPlasma");
int nS = backgroundPlasma["Z"].getLength();

cout << "nS  " << nS << endl;

Setting& diagnostics = cfg.lookup("diagnostics");
int nDensityChargeBins = diagnostics["densityChargeBins"].getLength();

cout << "nDensityChargeBins  " << nDensityChargeBins << endl;

densityChargeBins = new int[nDensityChargeBins];

background_Z = new int[nS];
background_amu = new double[nS];
background_flow = new double[nS];
maxDensity = new double[nS];
maxTemp_eV = new double[nS];

for(int i=0; i<nS; i++)
{
background_Z[i] = backgroundPlasma["Z"][i];
background_amu[i] = backgroundPlasma["amu"][i];
background_flow[i] = backgroundPlasma["flow"]["fractionOfThermalVelocity"][i];
maxDensity[i] = backgroundPlasma["density"]["max"][i];
maxTemp_eV[i] = backgroundPlasma["temp"]["max"][i];

cout << maxTemp_eV[i];
 }

    double x = cfg.lookup("impurityParticleSource.initialConditions.x_start");
    double y = cfg.lookup("impurityParticleSource.initialConditions.y_start");
    double z = cfg.lookup("impurityParticleSource.initialConditions.z_start");
    
    double Ex = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_x_start");
    double Ey = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_y_start");
    double Ez = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_z_start");
    
    double amu = cfg.lookup("impurityParticleSource.initialConditions.impurity_amu");
    double Z = cfg.lookup("impurityParticleSource.initialConditions.impurity_Z");

    double **SurfaceBins;
    double **SurfaceBinsCharge;
    double **SurfaceBinsEnergy;
    double **SurfaceBinsErosion;
    
    SurfaceBins = new double*[nY];
    SurfaceBinsCharge = new double*[nY];
    SurfaceBinsEnergy = new double*[nY];
    SurfaceBinsErosion = new double*[nY];

    SurfaceBins[0] = new double[nY*nZ];
    SurfaceBinsCharge[0] = new double[nY*nZ];
    SurfaceBinsEnergy[0] = new double[nY*nZ];
    SurfaceBinsErosion[0] = new double[nY*nZ];
            
    for(int i=0 ; i<nY ; i++)
    {
        SurfaceBins[i] = &SurfaceBins[0][i*nZ];
        SurfaceBinsCharge[i] = &SurfaceBinsCharge[0][i*nZ];
        SurfaceBinsEnergy[i] = &SurfaceBinsEnergy[0][i*nZ];
        SurfaceBinsErosion[i] = &SurfaceBinsErosion[0][i*nZ];               
        for(int j=0 ; j<nZ ; j++)
        {
            SurfaceBins[i][j] = 0;
            SurfaceBinsCharge[i][j] = 0;
            SurfaceBinsEnergy[i][j] = 0;
            SurfaceBinsErosion[i][j] = 0;
        }
    }
    
    double dt;
    double nPtsPerGyroOrbit = cfg.lookup("timeStep.nPtsPerGyroOrbit");
    dt = 1e-6/nPtsPerGyroOrbit;

    int nP = cfg.lookup("impurityParticleSource.nP");
    cout << "Number of particles: " << nP << endl;              
    long nParticles = nP;
    int nT = cfg.lookup("timeStep.nT");
    cout << "Number of time steps: " << nT << endl; 
    
    int surfaceIndexY;
    int surfaceIndexZ;

    Particle p1(x,y,z,Ex,Ey,Ez,Z,amu);

    std::cout << "nParticles: " << nParticles << std::endl;
#ifdef __HIPCC__
        thrust::host_vector<Particle> hostCudaParticleVector(nParticles,p1);
#else
        std::vector<Particle> hostCudaParticleVector(nParticles,p1);
#endif
        cpu_timer timer;

    std::cout << "Initial x position GPU: " << hostCudaParticleVector[1].x << "  " << hostCudaParticleVector[0].y << "  " << hostCudaParticleVector[0].z << "  " << hostCudaParticleVector[0].vx << "  " << hostCudaParticleVector[0].vy << "  " << hostCudaParticleVector[0].vz<< "  " << hostCudaParticleVector[0].Z << std::endl;
    
#ifdef __HIPCC__
    thrust::device_vector<Particle> deviceCudaParticleVector = hostCudaParticleVector;
#endif

    //std::uniform_real_distribution<float> dist(std::numeric_limits<float>::min(),std::numeric_limits<float>::max());
    std::uniform_real_distribution<float> dist(0,1e6);
        std::random_device rd;
        std::default_random_engine generator(rd());
    
#if USEIONIZATION > 0
    std::vector<float> seeds0(nP);
    std::generate( seeds0.begin(), seeds0.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
    thrust::device_vector<float> deviceSeeds0 = seeds0;
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds0.begin(), deviceCudaParticleVector.begin(), randInit(0) );
#else
    std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds0.begin(), hostCudaParticleVector.begin(), randInit(0) );
#endif
#endif

#if USERECOMBINATION > 0
        std::vector<float> seeds1(nP);
        std::generate( seeds1.begin(), seeds1.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds1 = seeds1;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds1.begin(), deviceCudaParticleVector.begin(), randInit(1) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds1.begin(), hostCudaParticleVector.begin(), randInit(1) );
#endif
#endif

#if USEPERPDIFFUSION > 0
        std::vector<float> seeds2(nP);
        std::generate( seeds2.begin(), seeds2.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds2 = seeds2;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds2.begin(), deviceCudaParticleVector.begin(), randInit(2) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds2.begin(), hostCudaParticleVector.begin(), randInit(2) );
#endif
#endif

#if USECOULOMBCOLLISIONS > 0
        std::vector<float> seeds3(nP),seeds4(nP),seeds5(nP);
        std::generate( seeds3.begin(), seeds3.end(), [&]() { return dist(generator); } );
    std::generate( seeds4.begin(), seeds4.end(), [&]() { return dist(generator); } );
    std::generate( seeds5.begin(), seeds5.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds3 = seeds3,deviceSeeds4 = seeds4,deviceSeeds5 = seeds5;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds3.begin(), deviceCudaParticleVector.begin(), randInit(3) );
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds4.begin(), deviceCudaParticleVector.begin(), randInit(4) );
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds5.begin(), deviceCudaParticleVector.begin(), randInit(5) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds3.begin(), hostCudaParticleVector.begin(), randInit(3) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds4.begin(), hostCudaParticleVector.begin(), randInit(4) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds5.begin(), hostCudaParticleVector.begin(), randInit(5) );
#endif
#endif

#if USESURFACEMODEL > 0
        std::vector<float> seeds6(nP);
        std::generate( seeds6.begin(), seeds6.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds6 = seeds6;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds6.begin(), deviceCudaParticleVector.begin(), randInit(6) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds6.begin(), hostCudaParticleVector.begin(), randInit(6) );
#endif
#endif
    
    cpu_times copyToDeviceTime = timer.elapsed();
    std::cout << "Initialize rand state and copyToDeviceTime: " << copyToDeviceTime.wall*1e-9 << '\n';
    for(int tt=0; tt< nT; tt++)
    {
#ifdef __HIPCC__
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), move_boris(dt) );
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), geometry_check(1.0) );
#if USEIONIZATION > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), ionize(dt) );
#endif
#if USERECOMBINATION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), crossFieldDiffusion(dt,perDiffusionCoeff_in));
#endif
#if USECOULOMBCOLLISIONS > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), coulombCollisions(dt) );
#endif
#if USETHERMALFORCE > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), thermalForce(dt) );
#endif
#else
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), move_boris(dt) );
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), geometry_check(1.0) );
#if USEIONIZATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), ionize(dt) );
#endif
#if USERECOMBINATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), crossFieldDiffusion(dt,perDiffusionCoeff_in));
#endif
#if USECOULOMBCOLLISIONS > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), coulombCollisions(dt) );
#endif
#if USETHERMALFORCE > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), thermalForce(dt) );
#endif
#endif
    }
    cpu_times ionizeTimeGPU = timer.elapsed();
    std::cout << "ionizeTimeGPU: " << ionizeTimeGPU.wall*1e-9 << '\n';
#ifdef __HIPCC__
    hostCudaParticleVector = deviceCudaParticleVector;
#endif

    for(int i=0; i < hostCudaParticleVector.size(); i++){
        //std::cout << " final pos" << hostCudaParticleVector[i].x << " " << hostCudaParticleVector[i].y << " " << hostCudaParticleVector[i].z << std::endl;
        if(hostCudaParticleVector[i].hitWall == 1){
        surfaceIndexY = int(floor((hostCudaParticleVector[i].y - yMin)/(yMax - yMin)*(nY) + 0.0f));
        surfaceIndexZ = int(floor((hostCudaParticleVector[i].z - zMin)/(zMax - zMin)*(nZ) + 0.0f));
        SurfaceBins[surfaceIndexY][surfaceIndexZ] +=  1.0 ;

        SurfaceBinsCharge[surfaceIndexY][surfaceIndexZ] += hostCudaParticleVector[i].Z ;
        SurfaceBinsEnergy[surfaceIndexY][surfaceIndexZ] += 0.5*hostCudaParticleVector[i].amu*1.6737236e-27*(hostCudaParticleVector[i].vx*hostCudaParticleVector[i].vx +  hostCudaParticleVector[i].vy*hostCudaParticleVector[i].vy+ hostCudaParticleVector[i].vz*hostCudaParticleVector[i].vz)/1.60217662e-19;
        }   
    }

    OUTPUT( outname,nY, nZ, SurfaceBins);
    OUTPUT( outnameCharge,nY, nZ, SurfaceBinsCharge);
    OUTPUT( outnameEnergy,nY, nZ, SurfaceBinsEnergy);
#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif

    cpu_times copyToHostTime = timer.elapsed();

    std::vector<Particle> particleVector(nParticles,p1);

    cpu_times createParticlesTimeCPU = timer.elapsed();
    std::cout << "createParticesTimeCPU: " << (createParticlesTimeCPU.wall-copyToHostTime.wall)*1e-9 << '\n';

    std::for_each( particleVector.begin(), particleVector.end(), move_boris(dt) );

    cpu_times moveTimeCPU = timer.elapsed();
    std::cout << "moveTimeCPU: " << (moveTimeCPU.wall-createParticlesTimeCPU.wall)*1e-9 << '\n';

    return 0;
}
