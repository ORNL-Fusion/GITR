#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include "h1.cuh"
#include <iomanip>
#include <cstdlib>
#include <libconfig.h++>
#include "boris.h"
#include "geometryCheck.h"
#include "ionize.h"
#include "recombine.h"
#include "crossFieldDiffusion.h"
#include "coulombCollisions.h"
#include "thermalForce.h"
#include "surfaceModel.h"
#include "interp2d.hpp"
#include "interpRateCoeff.hpp"
#include <algorithm>
#include <random>
#include "Particle.h"
#include "Boundary.h"
#include <boost/timer/timer.hpp>
#include <vector>
#include "io.hpp"
#include "testRoutine.h"
#include "testRoutineCuda.h"
#include "boundaryInit.h"

#ifdef __HIPCC__
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#endif

using namespace std;
using namespace libconfig;
using namespace boost::timer;

int main()
{

Config cfg,cfg_geom;

cfg.readFile("gitrInput.cfg");
cfg_geom.readFile("gitrGeometry.cfg");


// Background species info
double background_Z = cfg.lookup("backgroundPlasmaProfiles.Z");
double background_amu = cfg.lookup("backgroundPlasmaProfiles.amu");

#if BFIELD_INTERP == 0
int nR_Bfield = 1;
int nZ_Bfield = 1;
std::vector<double> bfieldGridr(nR_Bfield), bfieldGridz(nZ_Bfield);
std::vector<double> br(nR_Bfield*nZ_Bfield), bz(nR_Bfield*nZ_Bfield),bt(nR_Bfield*nZ_Bfield);
br[0] = cfg.lookup("backgroundPlasmaProfiles.Bfield.br");
bz[0] = cfg.lookup("backgroundPlasmaProfiles.Bfield.bz");
bt[0] = cfg.lookup("backgroundPlasmaProfiles.Bfield.bt");
#else
int nR_Bfield;
int nZ_Bfield;

int b1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.Bfield.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.Bfield.gridNzString"),nR_Bfield,nZ_Bfield);

std::vector<double> bfieldGridr(nR_Bfield), bfieldGridz(nZ_Bfield);
std::vector<double> br(nR_Bfield*nZ_Bfield), bz(nR_Bfield*nZ_Bfield),bt(nR_Bfield*nZ_Bfield);

int b2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.gridRString"), bfieldGridr);

int b3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.gridZString"), bfieldGridz);

int b4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.radialComponentString"), br);

int b5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.axialComponentString"), bz);

int b6 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.toroidalComponentString"), bt);
#endif
#ifdef __HIPCC__
    thrust::device_vector<double> deviceBfieldGridRVector = bfieldGridr;
    thrust::device_vector<double> deviceBfieldGridZVector = bfieldGridz;
    thrust::device_vector<double> deviceBfieldRVector = br;
    thrust::device_vector<double> deviceBfieldZVector = bz;
    thrust::device_vector<double> deviceBfieldTVector = bt;

    double * BfieldGridRDevicePointer = thrust::raw_pointer_cast(deviceBfieldGridRVector.data());
    double * BfieldGridZDevicePointer = thrust::raw_pointer_cast(deviceBfieldGridZVector.data());
    double * BfieldRDevicePointer = thrust::raw_pointer_cast(deviceBfieldRVector.data());
    double * BfieldZDevicePointer = thrust::raw_pointer_cast(deviceBfieldZVector.data());
    double * BfieldTDevicePointer = thrust::raw_pointer_cast(deviceBfieldTVector.data());
#endif
#if TEMP_INTERP == 0
int nR_Temp = 1;
int nZ_Temp = 1;
std::vector<double> TempGridr(nR_Temp), TempGridz(nZ_Temp);
std::vector<double> ti(nR_Temp*nZ_Temp), te(nR_Temp*nZ_Temp);
ti[0] = cfg.lookup("backgroundPlasmaProfiles.Temperature.ti");
te[0] = cfg.lookup("backgroundPlasmaProfiles.Temperature.te");
#else
int nR_Temp;
int nZ_Temp;

int t1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.Temperature.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.Temperature.gridNzString"),nR_Temp,nZ_Temp);
std::cout << "t1 " << nR_Temp << nZ_Temp << std::endl;
std::vector<double> TempGridr(nR_Temp), TempGridz(nZ_Temp);
std::vector<double> ti(nR_Temp*nZ_Temp), te(nR_Temp*nZ_Temp);

int t2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.gridRString"), TempGridr);

int t3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.gridZString"), TempGridz);
std::cout << "temperature import" << nZ_Temp << nR_Temp << std::endl;
int t4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.IonTempString"), ti);

int t5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.ElectronTempString"), te);
#endif

#if DENSITY_INTERP == 0
int nR_Dens = 1;
int nZ_Dens = 1;
std::vector<double> DensGridr(nR_Dens), DensGridz(nZ_Dens);
std::vector<double> ni(nR_Dens*nZ_Dens), ne(nR_Dens*nZ_Dens);
ni[0] = cfg.lookup("backgroundPlasmaProfiles.Temperature.ti");
ne[0] = cfg.lookup("backgroundPlasmaProfiles.Temperature.te");
#else
int nR_Dens;
int nZ_Dens;

int n1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.Density.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.Density.gridNzString"),nR_Dens,nZ_Dens);

std::vector<double> DensGridr(nR_Dens), DensGridz(nZ_Dens);
std::vector<double> ni(nR_Dens*nZ_Dens), ne(nR_Dens*nZ_Dens);

int n2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.gridRString"), DensGridr);

int n3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.gridZString"), DensGridz);

int n4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.IonDensityString"), ni);

int n5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.ElectronDensityString"), ne);
#endif
int nR_flowV;
int nZ_flowV;

int f1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridNzString"),nR_flowV,nZ_flowV);

std::vector<double> flowVGridr(nR_flowV), flowVGridz(nZ_flowV);
std::vector<double> flowVr(nR_flowV*nZ_flowV), flowVz(nR_flowV*nZ_flowV),flowVt(nR_flowV*nZ_flowV);

int f2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridRString"), flowVGridr);

int f3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridZString"), flowVGridz);

int f4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVrString"), flowVr);

int f5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVzString"), flowVz);

int f6 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVtString"), flowVt);

int nR_gradT;
int nZ_gradT;

int g1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.gradT.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.gradT.gridNzString"),nR_gradT,nZ_gradT);

std::vector<double> gradTGridr(nR_gradT), gradTGridz(nZ_gradT);
std::vector<double> gradTeR(nR_gradT*nZ_gradT), gradTeZ(nR_gradT*nZ_gradT),
    gradTiR(nR_gradT*nZ_gradT), gradTiZ(nR_gradT*nZ_gradT);

int g2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gridRString"), gradTGridr);

int g3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gridZString"), gradTGridz);

int g4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTiRString"), gradTiR);

int g5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTiZString"), gradTiZ);

int g6 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTeRString"), gradTeR);

int g7 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTeZString"), gradTeZ);
        std::cout << "thermal force numbers " << nR_gradT << " " << nZ_gradT << std::endl; 
     std::cout << "thermal grids r " << gradTGridr[0] << " " << gradTGridr[nR_gradT-1] << std::endl;
    std::cout << "thermal grids z " << gradTGridz[0] << " " << gradTGridz[nZ_gradT-1] << std::endl;

/*
string profilename("profiles.nc");
string densnxstring("n_x");
string densnzstring("n_z");
string densstring("ne");
string densgridxname("gridx");
string densgridzname("gridz");
int n_x;
int n_z;
int a1 = read_profileNs(profilename,densnxstring,densnzstring,n_x,n_z);
std::vector<double> gridx(n_x), gridz(n_z);
std::vector<double> dens(n_x*n_z);
int a2 = read_profiles(profilename,n_x,n_z,densgridxname, gridx,densgridzname, gridz,densstring, dens);
thrust::device_vector<double> device_dens = dens;
thrust::device_vector<double> device_gridx = gridx;
thrust::device_vector<double> device_gridz = gridz;
double interp_val1 = interp2dCombined(2.25, 0.0, -1.3,n_x,n_z, &gridx.front(), &gridz.front(), &dens.front());
std::cout << "interpolated value " << interp_val1 << std::endl;
std::vector<double> doubleVector(5,1.1);
thrust::device_vector<double> dd = doubleVector;

std::cout << "starting print loop" << std::endl;
std::for_each(doubleVector.begin(), doubleVector.end(), test_routine(2.25, 0.0, -1.3,n_x,n_z,&gridx.front(), &gridz.front(), &dens.front()) );
for (int i=0; i<5; i++)
{
        std::cout << "gridx: " << gridx[i] << std::endl;
}

double* gxptr2 = thrust::raw_pointer_cast(device_gridx.data());
double* gzptr2 = thrust::raw_pointer_cast(device_gridz.data());
double* dtptr2 = thrust::raw_pointer_cast(device_dens.data());
thrust::for_each(dd.begin(), dd.end(), test_routinecuda(2.25, 0.0, -1.3,n_x,n_z, gxptr2, gzptr2, dtptr2) );
thrust::host_vector<double> doubleVector2 = dd;
#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif
for (int i=0; i<5; i++)
{
        std::cout << "device doubleVector values: " << doubleVector2[i] << std::endl;
}

*/
int nCS_Ionize, nCS_Recombine;
int i0 = read_profileNs(cfg.lookup("impurityParticleSource.ionization.fileString"),
        cfg.lookup("impurityParticleSource.ionization.nChargeStateString"),
        cfg.lookup("impurityParticleSource.recombination.nChargeStateString"),
        nCS_Ionize, nCS_Recombine);
int nTemperaturesIonize;
int nDensitiesIonize;
int i1 = read_profileNs(cfg.lookup("impurityParticleSource.ionization.fileString"),
                    cfg.lookup("impurityParticleSource.ionization.DensGridString"),
         cfg.lookup("impurityParticleSource.ionization.TempGridString"),
         nDensitiesIonize,nTemperaturesIonize);

std::vector<double> rateCoeff_Ionization(nCS_Ionize*nTemperaturesIonize*nDensitiesIonize);
std::vector<double> gridTemperature_Ionization(nTemperaturesIonize),
    gridDensity_Ionization(nDensitiesIonize);

int    i2 = read_profiles(cfg.lookup("impurityParticleSource.ionization.fileString"),
        nTemperaturesIonize,nDensitiesIonize,
        cfg.lookup("impurityParticleSource.ionization.TempGridVarName"), 
        gridTemperature_Ionization,cfg.lookup("impurityParticleSource.ionization.DensGridVarName"),
        gridDensity_Ionization,
        cfg.lookup("impurityParticleSource.ionization.CoeffVarName"),
        rateCoeff_Ionization);
   

int nTemperaturesRecombine;
int nDensitiesRecombine;
int i3 = read_profileNs(cfg.lookup("impurityParticleSource.recombination.fileString"),
                    cfg.lookup("impurityParticleSource.recombination.DensGridString"),
         cfg.lookup("impurityParticleSource.recombination.TempGridString"),
         nDensitiesRecombine,nTemperaturesRecombine);

std::vector<double> rateCoeff_Recombination(nCS_Recombine*nTemperaturesRecombine*nDensitiesRecombine);
std::vector<double> gridTemperature_Recombination(nTemperaturesRecombine),
    gridDensity_Recombination(nDensitiesRecombine);

int    i4 = read_profiles(cfg.lookup("impurityParticleSource.recombination.fileString"),
        nTemperaturesRecombine,nDensitiesRecombine,
        cfg.lookup("impurityParticleSource.recombination.TempGridVarName"), 
        gridTemperature_Recombination,cfg.lookup("impurityParticleSource.recombination.DensGridVarName"),
        gridDensity_Recombination,
        cfg.lookup("impurityParticleSource.recombination.CoeffVarName"),
        rateCoeff_Recombination);

/*        std::cout << "Coeff vector print " << 
        rateCoeff_Ionization[0*nTemperaturesIonize*nDensitiesIonize+ 1*nTemperaturesIonize+ 0] << std::endl;
double RC1 = interpRateCoeff2d ( 0, 2.25, 0.0, -1.3,nR_Temp,nZ_Temp, &TempGridr.front(),
                      &TempGridz.front(),&te.front(),&DensGridr.front(),&DensGridz.front(), &ne.front(),nTemperaturesIonize,nDensitiesIonize,
       &gridTemperature_Ionization.front(),&gridDensity_Ionization.front(),&rateCoeff_Ionization.front() );
std::cout << "Interpolated RC " << RC1 << std::endl;
*/

char outname[] = "Deposition.m";
char outnameCharge[] = "Charge.m";
char outnameEnergy[] = "Energy.m";

//Geometry Definition
Setting& geom = cfg_geom.lookup("geom");
int nLines = geom["x1"].getLength();
std::cout << "Number of Geometric Objects Loaded: " << nLines << std::endl;

std::vector<Boundary> hostBoundaryVector(nLines+1);

for(int i=0 ; i<nLines ; i++)
    {
     hostBoundaryVector[i].x1 = geom["x1"][i];
     hostBoundaryVector[i].z1 = geom["z1"][i];
     hostBoundaryVector[i].x2 = geom["x2"][i];
     hostBoundaryVector[i].z2 = geom["z2"][i];
     hostBoundaryVector[i].Z = geom["Z"][i];
     hostBoundaryVector[i].slope_dzdx = geom["slope"][i];
     hostBoundaryVector[i].intercept_z = geom["intercept"][i];
     hostBoundaryVector[i].length = geom["length"][i];
    }   

hostBoundaryVector[nLines].Z = geom["Z"][nLines];
hostBoundaryVector[nLines].y1 = geom["y1"];
hostBoundaryVector[nLines].y2 = geom["y2"];
hostBoundaryVector[nLines].periodic = geom["periodic"];

std::for_each(hostBoundaryVector.begin(), hostBoundaryVector.end()-1, boundary_init(background_Z,background_amu,nR_Dens,nZ_Dens,&DensGridr.front(),&DensGridz.front(),&ni.front(),nR_Bfield,nZ_Bfield,&bfieldGridr.front(),&bfieldGridz.front(),&br.front(),&bz.front(), &bt.front(),
       nR_Temp,nZ_Temp,&TempGridr.front(),&TempGridz.front(),&ti.front() ));

#ifdef __HIPCC__
    thrust::device_vector<Boundary> deviceBoundaryVector = hostBoundaryVector;
    Boundary * BoundaryDevicePointer = thrust::raw_pointer_cast(deviceBoundaryVector.data());
#else
    std::vector<Boundary> * BoundaryHostPointer = &hostBoundaryVector;    
#endif
/*    // Volume definition

double xMinV = cfg.lookup("volumeDefinition.xMinV");
double xMaxV = cfg.lookup("volumeDefinition.xMaxV");
    // grid
int nXv = cfg.lookup("volumeDefinition.grid.nXv");
int nYv = cfg.lookup("volumeDefinition.grid.nYv");
int nZv = cfg.lookup("volumeDefinition.grid.nZv");

// Surface definition

double yMin = cfg.lookup("surfaceDefinition.yMin");
double yMax = cfg.lookup("surfaceDefinition.yMax");

double zMin = cfg.lookup("surfaceDefinition.zMin");
double zMax  = cfg.lookup("surfaceDefinition.zMax");


// Surface parameterization z = dz/dx * x + b

double surface_dz_dx  = cfg.lookup("surfaceDefinition.planeParameterization.surface_dz_dx");
double surface_zIntercept = cfg.lookup("surfaceDefinition.planeParameterization.surface_zIntercept");

// Constant B field value - only used when BfieldInterpolator_number = 0
double Bx_in = cfg.lookup("bField.Bx_in");
double By_in = cfg.lookup("bField.By_in");
double Bz_in = cfg.lookup("bField.Bz_in");
double connectionLength = cfg.lookup("bField.connectionLength");
*/
// Particle time stepping control

int ionization_nDtPerApply  = cfg.lookup("timeStep.ionization_nDtPerApply");
int collision_nDtPerApply  = cfg.lookup("timeStep.collision_nDtPerApply");
// Perp DiffusionCoeff - only used when Diffusion interpolator is = 0
double perpDiffusionCoeff = cfg.lookup("backgroundPlasmaProfiles.Diffusion.Dperp");

// Background species info
//double background_Z = cfg.lookup("backgroundPlasmaProfiles.Z");
//double background_amu = cfg.lookup("backgroundPlasmaProfiles.amu");
double *background_flow;
double *maxDensity;
double *maxTemp_eV;

#ifdef __HIPCC__
    cout<<"Using THRUST"<<endl;
#else
    cout<<"Not using THRUST"<<endl;
#endif

/*Setting& backgroundPlasma = cfg.lookup("backgroundPlasma");
int nS = backgroundPlasma["Z"].getLength();

Setting& diagnostics = cfg.lookup("diagnostics");


background_Z = new int[nS];
background_amu = new double[nS];
background_flow = new double[nS];
maxDensity = new double[nS];
maxTemp_eV = new double[nS];

for(int i=0; i<nS; i++)
{
background_Z[i] = backgroundPlasma["Z"][i];
background_amu[i] = backgroundPlasma["amu"][i];
background_flow[i] = backgroundPlasma["flow"]["fractionOfThermalVelocity"][i];
maxDensity[i] = backgroundPlasma["density"]["max"][i];
maxTemp_eV[i] = backgroundPlasma["temp"]["max"][i];
}
*/
    double x = cfg.lookup("impurityParticleSource.initialConditions.x_start");
    double y = cfg.lookup("impurityParticleSource.initialConditions.y_start");
    double z = cfg.lookup("impurityParticleSource.initialConditions.z_start");
    
    double Ex = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_x_start");
    double Ey = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_y_start");
    double Ez = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_z_start");
    
    double amu = cfg.lookup("impurityParticleSource.initialConditions.impurity_amu");
    double Z = cfg.lookup("impurityParticleSource.initialConditions.impurity_Z");
    double charge = cfg.lookup("impurityParticleSource.initialConditions.charge");
    /*
    double **SurfaceBins;
    double **SurfaceBinsCharge;
    double **SurfaceBinsEnergy;
    double **SurfaceBinsErosion;
    
    SurfaceBins = new double*[nY];
    SurfaceBinsCharge = new double*[nY];
    SurfaceBinsEnergy = new double*[nY];
    SurfaceBinsErosion = new double*[nY];

    SurfaceBins[0] = new double[nY*nZ];
    SurfaceBinsCharge[0] = new double[nY*nZ];
    SurfaceBinsEnergy[0] = new double[nY*nZ];
    SurfaceBinsErosion[0] = new double[nY*nZ];
            
    for(int i=0 ; i<nY ; i++)
    {
        SurfaceBins[i] = &SurfaceBins[0][i*nZ];
        SurfaceBinsCharge[i] = &SurfaceBinsCharge[0][i*nZ];
        SurfaceBinsEnergy[i] = &SurfaceBinsEnergy[0][i*nZ];
        SurfaceBinsErosion[i] = &SurfaceBinsErosion[0][i*nZ];               
        for(int j=0 ; j<nZ ; j++)
        {
            SurfaceBins[i][j] = 0;
            SurfaceBinsCharge[i][j] = 0;
            SurfaceBinsEnergy[i][j] = 0;
            SurfaceBinsErosion[i][j] = 0;
        }
    }
*/    
    double dt;
    double nPtsPerGyroOrbit = cfg.lookup("timeStep.nPtsPerGyroOrbit");
    dt = 1e-6/nPtsPerGyroOrbit;

    int nP = cfg.lookup("impurityParticleSource.nP");
    cout << "Number of particles: " << nP << endl;              
    long nParticles = nP;
    int nT = cfg.lookup("timeStep.nT");
    cout << "Number of time steps: " << nT << " With dt = " << dt << endl; 
    
//    int surfaceIndexY;
//    int surfaceIndexZ;
#if PARTICLE_SOURCE == 0
    Particle p1(x,y,z,Ex,Ey,Ez,Z,amu,charge);
#ifdef __HIPCC__
      thrust::host_vector<Particle> hostCudaParticleVector(nParticles,p1);
#else
        std::vector<Particle> hostCudaParticleVector(nParticles,p1);
#endif
#elif PARTICLE_SOURCE == 1
    double impurity_Z = cfg.lookup("impurityParticleSource.Z");
    int nImpurityBoundaries = 0;
    for (int i=0; i<nLines;i++)
    {
        if(hostBoundaryVector[i].Z == impurity_Z)
        {
            nImpurityBoundaries++;
        }
    }
    std::cout << "n Impurity Boundaries to launch from " << nImpurityBoundaries << std::endl;
    std::vector<int> boundaryIndex_ImpurityLaunch(nImpurityBoundaries);

    int count = 0;
    for (int i=0; i<nLines;i++)
    {
        if(hostBoundaryVector[i].Z == impurity_Z)
        {
            boundaryIndex_ImpurityLaunch[count] = i;
            count++;
            std::cout << "Boundary indices " << i << std::endl;
        }
    }
    
    int impuritiesPerBoundary = nP/nImpurityBoundaries;
#ifdef __HIPCC__
      thrust::host_vector<Particle> hostCudaParticleVector(nParticles);
#else
        std::vector<Particle> hostCudaParticleVector(nParticles);
#endif
   // Particle p1(0.0,0.0,0.0,0.0,0.0,0.0,0,0.0);
    for (int i=0; i< nImpurityBoundaries;i++)
    {
        //Set boundary interval, properties, and random number gen
        if (i==0)
        {
            x = 1.4290;
            z = -1.2540+0.01;
        }
        else
        {
            x = 1.3450;
            z = -1.3660+0.01;
        }
        for(int j=0; j<impuritiesPerBoundary; j++)
        {
            Particle p1(x,0.0,z,0.0,0.0,10,74,184.0,charge);
            hostCudaParticleVector[i*impuritiesPerBoundary + j] = p1;
        }
    }
#endif


#if GEOM_TRACE > 0       
            std::uniform_real_distribution<float> dist2(0,1);
            std::random_device rd2;
            std::cout << "Randomizing velocities to trace geometry. " << std::endl;
       
      for (int i=0 ; i<nParticles ; i++)
            {   double theta = dist2(rd2)*2*3.1415;
                double phi = dist2(rd2)*3.1415;
                double mag = 2e3;
                hostCudaParticleVector[i].vx = mag*cos(theta)*sin(phi);
                hostCudaParticleVector[i].vy = mag*sin(theta)*sin(phi);
                hostCudaParticleVector[i].vz = mag*cos(phi);
            }
#endif
#if PARTICLE_TRACKS > 0
double **positionHistoryX;
double **positionHistoryY;
double **positionHistoryZ;
double **velocityHistoryX;
double **velocityHistoryY;
double **velocityHistoryZ;
positionHistoryX = new double* [nP];
positionHistoryY = new double* [nP];
positionHistoryZ = new double* [nP];
velocityHistoryX = new double* [nP];
velocityHistoryY = new double* [nP];
velocityHistoryZ = new double* [nP];
positionHistoryX[0] = new double [nT*nP];
positionHistoryY[0] = new double [nT*nP];
positionHistoryZ[0] = new double [nT*nP];
velocityHistoryX[0] = new double [nT*nP];
velocityHistoryY[0] = new double [nT*nP];
velocityHistoryZ[0] = new double [nT*nP];
    for(int i=0 ; i<nP ; i++)
    {
        positionHistoryX[i] = &positionHistoryX[0][i*nT];
        positionHistoryY[i] = &positionHistoryY[0][i*nT];
        positionHistoryZ[i] = &positionHistoryZ[0][i*nT];
        velocityHistoryX[i] = &velocityHistoryX[0][i*nT];
        velocityHistoryY[i] = &velocityHistoryY[0][i*nT];
        velocityHistoryZ[i] = &velocityHistoryZ[0][i*nT];
        for(int j=0 ; j<nT ; j++)
        {
            positionHistoryX[i][j] = 0;
            positionHistoryY[i][j] = 0;
            positionHistoryZ[i][j] = 0;
            velocityHistoryX[i][j] = 0;
            velocityHistoryY[i][j] = 0;
            velocityHistoryZ[i][j] = 0;
        }
    }
#endif   
            cpu_timer timer;

#ifdef __HIPCC__
    thrust::device_vector<Particle> deviceCudaParticleVector = hostCudaParticleVector;
#endif

    std::uniform_real_distribution<float> dist(0,1e6);
        std::random_device rd;
        std::default_random_engine generator(rd());
    
#if USEIONIZATION > 0
    std::vector<float> seeds0(nP);
    std::generate( seeds0.begin(), seeds0.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
    thrust::device_vector<float> deviceSeeds0 = seeds0;
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds0.begin(), deviceCudaParticleVector.begin(), randInit(0) );
#else
    std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds0.begin(), hostCudaParticleVector.begin(), randInit(0) );
#endif
#endif

#if USERECOMBINATION > 0
        std::vector<float> seeds1(nP);
        std::generate( seeds1.begin(), seeds1.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds1 = seeds1;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds1.begin(), deviceCudaParticleVector.begin(), randInit(1) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds1.begin(), hostCudaParticleVector.begin(), randInit(1) );
#endif
#endif

#if USEPERPDIFFUSION > 0
        std::vector<float> seeds2(nP);
        std::generate( seeds2.begin(), seeds2.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds2 = seeds2;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds2.begin(), deviceCudaParticleVector.begin(), randInit(2) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds2.begin(), hostCudaParticleVector.begin(), randInit(2) );
#endif
#endif

#if USECOULOMBCOLLISIONS > 0
        std::vector<float> seeds3(nP),seeds4(nP),seeds5(nP);
        std::generate( seeds3.begin(), seeds3.end(), [&]() { return dist(generator); } );
    std::generate( seeds4.begin(), seeds4.end(), [&]() { return dist(generator); } );
    std::generate( seeds5.begin(), seeds5.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds3 = seeds3,deviceSeeds4 = seeds4,deviceSeeds5 = seeds5;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds3.begin(), deviceCudaParticleVector.begin(), randInit(3) );
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds4.begin(), deviceCudaParticleVector.begin(), randInit(4) );
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds5.begin(), deviceCudaParticleVector.begin(), randInit(5) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds3.begin(), hostCudaParticleVector.begin(), randInit(3) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds4.begin(), hostCudaParticleVector.begin(), randInit(4) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds5.begin(), hostCudaParticleVector.begin(), randInit(5) );
#endif
#endif

#if USESURFACEMODEL > 0
        std::vector<float> seeds6(nP);
        std::generate( seeds6.begin(), seeds6.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds6 = seeds6;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds6.begin(), deviceCudaParticleVector.begin(), randInit(6) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds6.begin(), hostCudaParticleVector.begin(), randInit(6) );
#endif
#endif
    
    cpu_times copyToDeviceTime = timer.elapsed();
    std::cout << "Initialize rand state and copyToDeviceTime: " << copyToDeviceTime.wall*1e-9 << '\n';
    for(int tt=0; tt< nT; tt++)
    {
#ifdef __HIPCC__
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), move_boris(dt,BoundaryDevicePointer, nLines,nR_Bfield,nZ_Bfield, BfieldGridRDevicePointer,BfieldGridZDevicePointer,
    BfieldRDevicePointer,BfieldZDevicePointer,BfieldTDevicePointer));
        try {
            thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), geometry_check(nLines,BoundaryDevicePointer) );
        }
        catch (thrust::system_error &e) {
            std::cerr << "Thrust system error: " << e.what() << std::endl;
            exit(-1);
        }
#if USEIONIZATION > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), ionize(dt) );
#endif
#if USERECOMBINATION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), crossFieldDiffusion(dt,perpDiffusionCoeff));
#endif
#if USECOULOMBCOLLISIONS > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), coulombCollisions(dt) );
#endif
#if USETHERMALFORCE > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), thermalForce(dt) );
#endif
#else
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), move_boris(dt,hostBoundaryVector,nLines, nR_Bfield,nZ_Bfield, &bfieldGridr.front(),&bfieldGridz.front(),
                &br.front(),&bz.front(),&bt.front()));
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), geometry_check(nLines,hostBoundaryVector) );
#if USEIONIZATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), ionize(dt,
                nR_Dens,nZ_Dens,&DensGridr.front(),&DensGridz.front(),&ne.front(),
                nR_Temp,nZ_Temp,&TempGridr.front(),&TempGridz.front(),&te.front(),
                nTemperaturesIonize, nDensitiesIonize, &gridTemperature_Ionization.front(),
               &gridDensity_Ionization.front(), &rateCoeff_Ionization.front() ) );
#endif
#if USERECOMBINATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
    //std::cout<< "Perp diffusion loop " << perpDiffusionCoeff << std::endl;
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), crossFieldDiffusion(dt,perpDiffusionCoeff,nR_Bfield,nZ_Bfield, &bfieldGridr.front(),&bfieldGridz.front(),
                                    &br.front(),&bz.front(),&bt.front()));
#endif
#if USECOULOMBCOLLISIONS > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), coulombCollisions(dt,nR_flowV,nZ_flowV,&flowVGridr.front(),&flowVGridz.front(),&flowVr.front(),&flowVz.front(),
                    &flowVt.front(),
                nR_Dens,nZ_Dens,&DensGridr.front(),&DensGridz.front(),&ne.front(),
                nR_Temp,nZ_Temp,&TempGridr.front(),&TempGridz.front(),&te.front(),
                background_Z,background_amu,nR_Bfield,nZ_Bfield, &bfieldGridr.front(),
                &bfieldGridz.front(),&br.front(),&bz.front(),&bt.front()));
#endif
#if USETHERMALFORCE > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), thermalForce(dt,background_amu,nR_gradT,nZ_gradT,&gradTGridr.front(),&gradTGridz.front(),
                    &gradTiR.front(),&gradTiZ.front(),&gradTeR.front(),&gradTeZ.front() ) );
#endif
#if PARTICLE_TRACKS >0
        for(int i=0;i<nP;i++)
        {
            positionHistoryX[i][tt] = hostCudaParticleVector[i].xprevious;
            positionHistoryY[i][tt] = hostCudaParticleVector[i].yprevious;
            positionHistoryZ[i][tt] = hostCudaParticleVector[i].zprevious;
            velocityHistoryX[i][tt] = hostCudaParticleVector[i].vx;
            velocityHistoryY[i][tt] = hostCudaParticleVector[i].vy;
            velocityHistoryZ[i][tt] = hostCudaParticleVector[i].vz;
        }
#endif
#endif
    }
    cpu_times ionizeTimeGPU = timer.elapsed();
    std::cout << "Particle Moving Time: " << ionizeTimeGPU.wall*1e-9 << '\n';

#ifdef __HIPCC__
    hostCudaParticleVector = deviceCudaParticleVector;
#endif

    for(int i=0; i < hostCudaParticleVector.size(); i++){
       //std::cout << " final pos" <<  i << " " <<hostCudaParticleVector[i].x << " " << hostCudaParticleVector[i].y << " " << hostCudaParticleVector[i].z << std::endl;
        /*if(hostCudaParticleVector[i].hitWall == 1){
        surfaceIndexY = int(floor((hostCudaParticleVector[i].y - yMin)/(yMax - yMin)*(nY) + 0.0f));
        surfaceIndexZ = int(floor((hostCudaParticleVector[i].z - zMin)/(zMax - zMin)*(nZ) + 0.0f));
        SurfaceBins[surfaceIndexY][surfaceIndexZ] +=  1.0 ;

        SurfaceBinsCharge[surfaceIndexY][surfaceIndexZ] += hostCudaParticleVector[i].Z ;
        SurfaceBinsEnergy[surfaceIndexY][surfaceIndexZ] += 0.5*hostCudaParticleVector[i].amu*1.6737236e-27*(hostCudaParticleVector[i].vx*hostCudaParticleVector[i].vx +  hostCudaParticleVector[i].vy*hostCudaParticleVector[i].vy+ hostCudaParticleVector[i].vz*hostCudaParticleVector[i].vz)/1.60217662e-19;
        } */ 
    }

//    OUTPUT( outname,nY, nZ, SurfaceBins);
//    OUTPUT( outnameCharge,nY, nZ, SurfaceBinsCharge);
//    OUTPUT( outnameEnergy,nY, nZ, SurfaceBinsEnergy);

    ofstream outfile2;
    outfile2.open ("positions.m");
    for(int i=1 ; i<=nP ; i++)
      {
        outfile2 << "Pos( " << i<< ",:) = [ " ;
        outfile2 << hostCudaParticleVector[i-1].x << " " << hostCudaParticleVector[i-1].y << " " << hostCudaParticleVector[i-1].z << " ];" << std::endl;
      }
       outfile2.close();
#if PARTICLE_TRACKS > 0
char outnameX[] = "positionHistoryX.m";
OUTPUT( outnameX,nP, nT, positionHistoryX);
char outnameY[] = "positionHistoryY.m";
OUTPUT( outnameY,nP, nT, positionHistoryY);
char outnameZ[] = "positionHistoryZ.m";
OUTPUT( outnameZ,nP, nT, positionHistoryZ);
char outnameVX[] = "velocityHistoryX.m";
OUTPUT( outnameVX,nP, nT,velocityHistoryX);
char outnameVY[] = "velocityHistoryY.m";
OUTPUT( outnameVY,nP, nT, velocityHistoryY);
char outnameVZ[] = "velocityHistoryZ.m";
OUTPUT( outnameVZ,nP, nT, velocityHistoryZ);
#endif

#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif

    cpu_times copyToHostTime = timer.elapsed();

    cpu_times createParticlesTimeCPU = timer.elapsed();
    std::cout << "Copy to host, bin and output time: " << (createParticlesTimeCPU.wall-copyToHostTime.wall)*1e-9 << '\n';
    return 0;
}
