#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include "h1.cuh"
#include <iomanip>
#include <cstdlib>
#include <libconfig.h++>
#include "boris.h"
#include "geometryCheck.h"
#include "ionize.h"
#include "recombine.h"
#include "crossFieldDiffusion.h"
#include "coulombCollisions.h"
#include "thermalForce.h"
#include "surfaceModel.h"
#include "interp2d.hpp"
#include "interpRateCoeff.hpp"
#include <algorithm>
#include <random>
#include "Particle.h"
#include "Boundary.h"
#include <boost/timer/timer.hpp>
#include "boost/filesystem.hpp"
#include <vector>
#include "io.hpp"
#include "testRoutine.h"
#include "testRoutineCuda.h"
#include "boundaryInit.h"
#include <netcdf>

#ifdef __HIPCC__
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#endif

using namespace std;
using namespace libconfig;
using namespace boost::timer;
using namespace netCDF;
using namespace exceptions;

int main()
{

Config cfg,cfg_geom;

cfg.readFile("gitrInput.cfg");
cfg_geom.readFile("gitrGeometry.cfg");


// Background species info
double background_Z = cfg.lookup("backgroundPlasmaProfiles.Z");
double background_amu = cfg.lookup("backgroundPlasmaProfiles.amu");

std::string outnameBfieldR = "BfieldR.m";
std::string outnameBfieldZ = "BfieldZ.m";
std::string outnameBfieldT = "BfieldT.m";
std::string outnameGridR = "gridR.m";
std::string outnameGridZ = "gridZ.m";
std::string profiles_folder = "profiles";
#if BFIELD_INTERP == 0
int nR_Bfield = 1;
int nZ_Bfield = 1;
std::vector<double> bfieldGridr(nR_Bfield), bfieldGridz(nZ_Bfield);
std::vector<double> br(nR_Bfield*nZ_Bfield), bz(nR_Bfield*nZ_Bfield),bt(nR_Bfield*nZ_Bfield);
br[0] = cfg.lookup("backgroundPlasmaProfiles.Bfield.br");
bz[0] = cfg.lookup("backgroundPlasmaProfiles.Bfield.bz");
bt[0] = cfg.lookup("backgroundPlasmaProfiles.Bfield.bt");
#else
int nR_Bfield;
int nZ_Bfield;

int b1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.Bfield.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.Bfield.gridNzString"),nR_Bfield,nZ_Bfield);

std::vector<double> bfieldGridr(nR_Bfield), bfieldGridz(nZ_Bfield);
std::vector<double> br(nR_Bfield*nZ_Bfield), bz(nR_Bfield*nZ_Bfield),bt(nR_Bfield*nZ_Bfield);

int b2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.gridRString"), bfieldGridr);

int b3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.gridZString"), bfieldGridz);

int b4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.radialComponentString"), br);

int b5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.axialComponentString"), bz);

int b6 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Bfield.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Bfield.toroidalComponentString"), bt);

OUTPUT1d(profiles_folder,outnameGridR, nR_Bfield, &bfieldGridr.front());
OUTPUT1d(profiles_folder,outnameGridZ, nZ_Bfield, &bfieldGridz.front());
OUTPUT2d(profiles_folder,outnameBfieldR, nR_Bfield, nZ_Bfield, &br.front());
OUTPUT2d(profiles_folder,outnameBfieldZ, nR_Bfield, nZ_Bfield, &bz.front());
OUTPUT2d(profiles_folder,outnameBfieldT, nR_Bfield, nZ_Bfield, &bt.front());
#endif
#ifdef __HIPCC__
    thrust::device_vector<double> deviceBfieldGridRVector = bfieldGridr;
    thrust::device_vector<double> deviceBfieldGridZVector = bfieldGridz;
    thrust::device_vector<double> deviceBfieldRVector = br;
    thrust::device_vector<double> deviceBfieldZVector = bz;
    thrust::device_vector<double> deviceBfieldTVector = bt;

    double * BfieldGridRDevicePointer = thrust::raw_pointer_cast(deviceBfieldGridRVector.data());
    double * BfieldGridZDevicePointer = thrust::raw_pointer_cast(deviceBfieldGridZVector.data());
    double * BfieldRDevicePointer = thrust::raw_pointer_cast(deviceBfieldRVector.data());
    double * BfieldZDevicePointer = thrust::raw_pointer_cast(deviceBfieldZVector.data());
    double * BfieldTDevicePointer = thrust::raw_pointer_cast(deviceBfieldTVector.data());
#endif
#if TEMP_INTERP == 0
int nR_Temp = 1;
int nZ_Temp = 1;
std::vector<double> TempGridr(nR_Temp), TempGridz(nZ_Temp);
std::vector<double> ti(nR_Temp*nZ_Temp), te(nR_Temp*nZ_Temp);
ti[0] = cfg.lookup("backgroundPlasmaProfiles.Temperature.ti");
te[0] = cfg.lookup("backgroundPlasmaProfiles.Temperature.te");
#else
int nR_Temp;
int nZ_Temp;

int t1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.Temperature.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.Temperature.gridNzString"),nR_Temp,nZ_Temp);
std::cout << "t1 " << nR_Temp << nZ_Temp << std::endl;
std::vector<double> TempGridr(nR_Temp), TempGridz(nZ_Temp);
std::vector<double> ti(nR_Temp*nZ_Temp), te(nR_Temp*nZ_Temp);

int t2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.gridRString"), TempGridr);

int t3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.gridZString"), TempGridz);
std::cout << "temperature import" << nZ_Temp << nR_Temp << std::endl;
int t4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.IonTempString"), ti);

int t5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Temperature.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Temperature.ElectronTempString"), te);
std::string outnameTi = "ti.m";
std::string outnameTe = "te.m";
OUTPUT2d(profiles_folder,outnameTi, nR_Temp, nZ_Temp, &ti.front());
OUTPUT2d(profiles_folder,outnameTe, nR_Temp, nZ_Temp, &te.front());
#endif

#ifdef __HIPCC__
    thrust::device_vector<double> deviceTempGridRVector = TempGridr;
    thrust::device_vector<double> deviceTempGridZVector = TempGridz;
    thrust::device_vector<double> deviceTiVector = ti;
    thrust::device_vector<double> deviceTeVector = te;
    double * TempGridRDevicePointer = thrust::raw_pointer_cast(deviceTempGridRVector.data());
    double * TempGridZDevicePointer = thrust::raw_pointer_cast(deviceTempGridZVector.data());
    double * TiDevicePointer = thrust::raw_pointer_cast(deviceTiVector.data());
    double * TeDevicePointer = thrust::raw_pointer_cast(deviceTeVector.data());
#endif


#if DENSITY_INTERP == 0
int nR_Dens = 1;
int nZ_Dens = 1;
std::vector<double> DensGridr(nR_Dens), DensGridz(nZ_Dens);
std::vector<double> ni(nR_Dens*nZ_Dens), ne(nR_Dens*nZ_Dens);
ni[0] = cfg.lookup("backgroundPlasmaProfiles.Density.ni");
ne[0] = cfg.lookup("backgroundPlasmaProfiles.Density.ne");
#else
int nR_Dens;
int nZ_Dens;

int n1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.Density.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.Density.gridNzString"),nR_Dens,nZ_Dens);

std::vector<double> DensGridr(nR_Dens), DensGridz(nZ_Dens);
std::vector<double> ni(nR_Dens*nZ_Dens), ne(nR_Dens*nZ_Dens);

int n2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.gridRString"), DensGridr);

int n3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.gridZString"), DensGridz);

int n4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.IonDensityString"), ni);

int n5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.Density.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.Density.ElectronDensityString"), ne);
std::string outnameNi = "ni.m";
std::string outnameNe = "ne.m";
OUTPUT2d(profiles_folder,outnameNi, nR_Dens, nZ_Dens, &ni.front());
OUTPUT2d(profiles_folder,outnameNe, nR_Dens, nZ_Dens, &ne.front());
#endif

#ifdef __HIPCC__
    thrust::device_vector<double> deviceDensGridRVector = DensGridr;
    thrust::device_vector<double> deviceDensGridZVector = DensGridz;
    thrust::device_vector<double> deviceNiVector = ni;
    thrust::device_vector<double> deviceNeVector = ne;
    double * DensGridRDevicePointer = thrust::raw_pointer_cast(deviceDensGridRVector.data());
    double * DensGridZDevicePointer = thrust::raw_pointer_cast(deviceDensGridZVector.data());
    double * NiDevicePointer = thrust::raw_pointer_cast(deviceNiVector.data());
    double * NeDevicePointer = thrust::raw_pointer_cast(deviceNeVector.data());
#endif

#if FLOWV_INTERP == 0
int nR_flowV = 1;
int nZ_flowV = 1;
std::vector<double> flowVGridr(nR_flowV), flowVGridz(nZ_flowV);
std::vector<double> flowVr(nR_flowV*nZ_flowV), flowVz(nR_flowV*nZ_flowV),flowVt(nR_flowV*nZ_flowV);
flowVr[0] = cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVr");
flowVz[0] = cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVz");
#else
int nR_flowV;
int nZ_flowV;

int f1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridNzString"),nR_flowV,nZ_flowV);

std::vector<double> flowVGridr(nR_flowV), flowVGridz(nZ_flowV);
std::vector<double> flowVr(nR_flowV*nZ_flowV), flowVz(nR_flowV*nZ_flowV),flowVt(nR_flowV*nZ_flowV);

int f2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridRString"), flowVGridr);

int f3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.gridZString"), flowVGridz);

int f4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVrString"), flowVr);

int f5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVzString"), flowVz);

int f6 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.FlowVelocity.flowVtString"), flowVt);

std::string outnameFlowVr = "flowVr.m";
std::string outnameFlowVz = "flowVz.m";
std::string outnameFlowVt = "flowVt.m";
OUTPUT2d(profiles_folder,outnameFlowVr, nR_flowV, nZ_flowV, &flowVr.front());
OUTPUT2d(profiles_folder,outnameFlowVz, nR_flowV, nZ_flowV, &flowVz.front());
OUTPUT2d(profiles_folder,outnameFlowVt, nR_flowV, nZ_flowV, &flowVt.front());
#endif

#ifdef __HIPCC__
    thrust::device_vector<double> deviceFlowVGridRVector = flowVGridr;
    thrust::device_vector<double> deviceFlowVGridZVector = flowVGridz;
    thrust::device_vector<double> deviceFlowVrVector = flowVr;
    thrust::device_vector<double> deviceFlowVzVector = flowVz;
    thrust::device_vector<double> deviceFlowVtVector = flowVt;
    double * FlowVGridRDevicePointer = thrust::raw_pointer_cast(deviceFlowVGridRVector.data());
    double * FlowVGridZDevicePointer = thrust::raw_pointer_cast(deviceFlowVGridZVector.data());
    double * FlowVrDevicePointer = thrust::raw_pointer_cast(deviceFlowVrVector.data());
    double * FlowVzDevicePointer = thrust::raw_pointer_cast(deviceFlowVzVector.data());
    double * FlowVtDevicePointer = thrust::raw_pointer_cast(deviceFlowVtVector.data());
#endif

#if GRADT_INTERP == 0
int nR_gradT = 1;
int nZ_gradT = 1;
std::vector<double> gradTGridr(nR_gradT), gradTGridz(nZ_gradT);
std::vector<double> gradTeR(nR_gradT*nZ_gradT), gradTeZ(nR_gradT*nZ_gradT),
    gradTiR(nR_gradT*nZ_gradT), gradTiZ(nR_gradT*nZ_gradT);
gradTeR[0] = cfg.lookup("backgroundPlasmaProfiles.gradT.gradTeR");
gradTeZ[0] = cfg.lookup("backgroundPlasmaProfiles.gradT.gradTeZ");
gradTiR[0] = cfg.lookup("backgroundPlasmaProfiles.gradT.gradTiR");
gradTiZ[0] = cfg.lookup("backgroundPlasmaProfiles.gradT.gradTiZ");
#else
int nR_gradT;
int nZ_gradT;

int g1 = read_profileNs(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
            cfg.lookup("backgroundPlasmaProfiles.gradT.gridNrString"),
            cfg.lookup("backgroundPlasmaProfiles.gradT.gridNzString"),nR_gradT,nZ_gradT);

std::vector<double> gradTGridr(nR_gradT), gradTGridz(nZ_gradT);
std::vector<double> gradTeR(nR_gradT*nZ_gradT), gradTeZ(nR_gradT*nZ_gradT),
    gradTiR(nR_gradT*nZ_gradT), gradTiZ(nR_gradT*nZ_gradT);

int g2 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gridRString"), gradTGridr);

int g3 = read_profile1d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gridZString"), gradTGridz);

int g4 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTiRString"), gradTiR);

int g5 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTiZString"), gradTiZ);

int g6 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTeRString"), gradTeR);

int g7 = read_profile2d(cfg.lookup("backgroundPlasmaProfiles.gradT.fileString"),
        cfg.lookup("backgroundPlasmaProfiles.gradT.gradTeZString"), gradTeZ);
std::string outnameGradTiR = "gradTiR.m";
std::string outnameGradTiZ = "gradTiZ.m";
std::string outnameGradTeR = "gradTeR.m";
std::string outnameGradTeZ = "gradTeZ.m";
OUTPUT2d(profiles_folder,outnameGradTiR, nR_gradT, nZ_gradT, &gradTiR.front());
OUTPUT2d(profiles_folder,outnameGradTiZ, nR_gradT, nZ_gradT, &gradTiZ.front());
OUTPUT2d(profiles_folder,outnameGradTeR, nR_gradT, nZ_gradT, &gradTeR.front());
OUTPUT2d(profiles_folder,outnameGradTeZ, nR_gradT, nZ_gradT, &gradTeZ.front());
#endif

#ifdef __HIPCC__
    thrust::device_vector<double> deviceGradTGridRVector = gradTGridr;
    thrust::device_vector<double> deviceGradTGridZVector = gradTGridz;
    thrust::device_vector<double> deviceGradTiRVector = gradTiR;
    thrust::device_vector<double> deviceGradTiZVector = gradTiZ;
    thrust::device_vector<double> deviceGradTeRVector = gradTeR;
    thrust::device_vector<double> deviceGradTeZVector = gradTeZ;
    double * GradTGridRDevicePointer = thrust::raw_pointer_cast(deviceGradTGridRVector.data());
    double * GradTGridZDevicePointer = thrust::raw_pointer_cast(deviceGradTGridZVector.data());
    double * GradTiRDevicePointer = thrust::raw_pointer_cast(deviceGradTiRVector.data());
    double * GradTiZDevicePointer = thrust::raw_pointer_cast(deviceGradTiZVector.data());
    double * GradTeRDevicePointer = thrust::raw_pointer_cast(deviceGradTeRVector.data());
    double * GradTeZDevicePointer = thrust::raw_pointer_cast(deviceGradTeZVector.data());
#endif

int nCS_Ionize, nCS_Recombine;
int i0 = read_profileNs(cfg.lookup("impurityParticleSource.ionization.fileString"),
        cfg.lookup("impurityParticleSource.ionization.nChargeStateString"),
        cfg.lookup("impurityParticleSource.recombination.nChargeStateString"),
        nCS_Ionize, nCS_Recombine);
int nTemperaturesIonize;
int nDensitiesIonize;
int i1 = read_profileNs(cfg.lookup("impurityParticleSource.ionization.fileString"),
                    cfg.lookup("impurityParticleSource.ionization.DensGridString"),
         cfg.lookup("impurityParticleSource.ionization.TempGridString"),
         nDensitiesIonize,nTemperaturesIonize);

std::vector<double> rateCoeff_Ionization(nCS_Ionize*nTemperaturesIonize*nDensitiesIonize);
std::vector<double> gridTemperature_Ionization(nTemperaturesIonize),
    gridDensity_Ionization(nDensitiesIonize);

int    i2 = read_profiles(cfg.lookup("impurityParticleSource.ionization.fileString"),
        nTemperaturesIonize,nDensitiesIonize,
        cfg.lookup("impurityParticleSource.ionization.TempGridVarName"), 
        gridTemperature_Ionization,cfg.lookup("impurityParticleSource.ionization.DensGridVarName"),
        gridDensity_Ionization,
        cfg.lookup("impurityParticleSource.ionization.CoeffVarName"),
        rateCoeff_Ionization);
   

int nTemperaturesRecombine;
int nDensitiesRecombine;
int i3 = read_profileNs(cfg.lookup("impurityParticleSource.recombination.fileString"),
                    cfg.lookup("impurityParticleSource.recombination.DensGridString"),
         cfg.lookup("impurityParticleSource.recombination.TempGridString"),
         nDensitiesRecombine,nTemperaturesRecombine);

std::vector<double> rateCoeff_Recombination(nCS_Recombine*nTemperaturesRecombine*nDensitiesRecombine);
std::vector<double> gridTemperature_Recombination(nTemperaturesRecombine),
    gridDensity_Recombination(nDensitiesRecombine);

int    i4 = read_profiles(cfg.lookup("impurityParticleSource.recombination.fileString"),
        nTemperaturesRecombine,nDensitiesRecombine,
        cfg.lookup("impurityParticleSource.recombination.TempGridVarName"), 
        gridTemperature_Recombination,cfg.lookup("impurityParticleSource.recombination.DensGridVarName"),
        gridDensity_Recombination,
        cfg.lookup("impurityParticleSource.recombination.CoeffVarName"),
        rateCoeff_Recombination);

#ifdef __HIPCC__
    thrust::device_vector<double> deviceTemperatureGridIonizationVector = gridTemperature_Ionization;
    thrust::device_vector<double> deviceDensityGridIonizationVector = gridDensity_Ionization;
    thrust::device_vector<double> deviceTemperatureGridRecombinationVector = gridTemperature_Recombination;
    thrust::device_vector<double> deviceDensityGridRecombinationVector = gridDensity_Recombination;
    thrust::device_vector<double> deviceRateCoeffIonizationVector = rateCoeff_Ionization;
    thrust::device_vector<double> deviceRateCoeffRecombinationVector = rateCoeff_Recombination;
    double * IonizationTemperatureGridDevicePointer = thrust::raw_pointer_cast(deviceTemperatureGridIonizationVector.data());
    double * RecombinationTemperatureGridDevicePointer = thrust::raw_pointer_cast(deviceTemperatureGridRecombinationVector.data());
    double * IonizationDensityGridDevicePointer = thrust::raw_pointer_cast(deviceDensityGridIonizationVector.data());
    double * RecombinationDensityGridDevicePointer = thrust::raw_pointer_cast(deviceDensityGridRecombinationVector.data());
    double * IonizationRateCoeffDevicePointer = thrust::raw_pointer_cast(deviceRateCoeffIonizationVector.data());
    double * RecombinationRateCoeffDevicePointer = thrust::raw_pointer_cast(deviceRateCoeffRecombinationVector.data());
#endif

char outname[] = "Deposition.m";
char outnameCharge[] = "Charge.m";
char outnameEnergy[] = "Energy.m";

//Geometry Definition
Setting& geom = cfg_geom.lookup("geom");
int nLines = geom["x1"].getLength();
std::cout << "Number of Geometric Objects Loaded: " << nLines << std::endl;

std::vector<Boundary> hostBoundaryVector(nLines+1);

std::string geom_outname = "geom.m";
std::string geom_folder = "geometry";
       ofstream outfile;
                       //Output
        boost::filesystem::path dir(geom_folder);

       if(!(boost::filesystem::exists(dir)))
       {
       // std::cout<<"Doesn't Exists"<<std::endl;
        if (boost::filesystem::create_directory(dir))
        {
        //std::cout << " Successfully Created " << std::endl;
        }
       }
       std::string full_path = geom_folder + "/" + geom_outname;
      outfile.open (full_path );

for(int i=0 ; i<nLines ; i++)
    {
     hostBoundaryVector[i].x1 = geom["x1"][i];
     hostBoundaryVector[i].z1 = geom["z1"][i];
     hostBoundaryVector[i].x2 = geom["x2"][i];
     hostBoundaryVector[i].z2 = geom["z2"][i];
     hostBoundaryVector[i].Z = geom["Z"][i];
     hostBoundaryVector[i].slope_dzdx = geom["slope"][i];
     hostBoundaryVector[i].intercept_z = geom["intercept"][i];
     hostBoundaryVector[i].length = geom["length"][i];

    outfile << "geom(" << i+1 << ",:) = ["<<hostBoundaryVector[i].x1 << ", " <<hostBoundaryVector[i].z1 << ", " <<
        hostBoundaryVector[i].x2 << ", " << hostBoundaryVector[i].z2 << ", " <<
        hostBoundaryVector[i].slope_dzdx << ", " << hostBoundaryVector[i].intercept_z << ", " <<
        hostBoundaryVector[i].length << ", " << hostBoundaryVector[i].Z << "];" << std::endl;
    }   

outfile.close();

hostBoundaryVector[nLines].Z = geom["Z"][nLines];
hostBoundaryVector[nLines].y1 = geom["y1"];
hostBoundaryVector[nLines].y2 = geom["y2"];
hostBoundaryVector[nLines].periodic = geom["periodic"];
std::cout << "Starting Boundary Init..." << std::endl;
std::for_each(hostBoundaryVector.begin(), hostBoundaryVector.end()-1, boundary_init(background_Z,background_amu,nR_Dens,nZ_Dens,&DensGridr.front(),&DensGridz.front(),&ni.front(),nR_Bfield,nZ_Bfield,&bfieldGridr.front(),&bfieldGridz.front(),&br.front(),&bz.front(), &bt.front(),
       nR_Temp,nZ_Temp,&TempGridr.front(),&TempGridz.front(),&ti.front() ));

#ifdef __HIPCC__
    thrust::device_vector<Boundary> deviceBoundaryVector = hostBoundaryVector;
    Boundary * BoundaryDevicePointer = thrust::raw_pointer_cast(deviceBoundaryVector.data());
#else
    std::vector<Boundary> * BoundaryHostPointer = &hostBoundaryVector;    
#endif
//Efield
    
std::vector<double> Efieldr(nR_Bfield*nZ_Bfield), Efieldz(nR_Bfield*nZ_Bfield),Efieldt(nR_Bfield*nZ_Bfield),minDist(nR_Bfield*nZ_Bfield);
double thisE[3] = {0.0,0.0,0.0};

for(int i=0;i<nR_Bfield;i++)
{
    for(int j=0;j<nZ_Bfield;j++)
    {
        minDist[(nR_Bfield - 1 -i)*nZ_Bfield+(nZ_Bfield -1-j)] = getE ( bfieldGridr[i], 0.0, bfieldGridz[j],  thisE, hostBoundaryVector,nLines );
        //std::cout << "point " << bfieldGridr[i] << " " << bfieldGridz[j] << " dist" << minDist[(i)*nZ_Bfield+(j)] << std::endl;
        Efieldr[i*nZ_Bfield+j] = thisE[0];
        Efieldz[i*nZ_Bfield+j] = thisE[2];
        Efieldt[i*nZ_Bfield+j] = thisE[1];
    }
}
for(int i=0;i<nR_Bfield;i++)
{
    for(int j=0;j<nZ_Bfield;j++)
    {
       // std::cout << "point " << bfieldGridr[i] << " " << bfieldGridz[j] << " dist" << minDist[(i)+(j)*nR_Bfield] << std::endl;
    }
}
std::string outnameEfieldR = "EfieldR.m";
std::string outnameEfieldZ = "EfieldZ.m";
std::string outnameEfieldT = "EfieldT.m";
std::string outnameMinDist = "DistToSurface.m";
OUTPUT2d(profiles_folder,outnameEfieldR, nR_Bfield, nZ_Bfield, &Efieldr.front());
OUTPUT2d(profiles_folder,outnameEfieldZ, nR_Bfield, nZ_Bfield, &Efieldz.front());
OUTPUT2d(profiles_folder,outnameEfieldT, nR_Bfield, nZ_Bfield, &Efieldt.front());
OUTPUT2d(profiles_folder,outnameMinDist, nR_Bfield, nZ_Bfield, &minDist.front());

// Particle time stepping control

int ionization_nDtPerApply  = cfg.lookup("timeStep.ionization_nDtPerApply");
int collision_nDtPerApply  = cfg.lookup("timeStep.collision_nDtPerApply");
// Perp DiffusionCoeff - only used when Diffusion interpolator is = 0
double perpDiffusionCoeff = cfg.lookup("backgroundPlasmaProfiles.Diffusion.Dperp");

// Background species info
//double background_Z = cfg.lookup("backgroundPlasmaProfiles.Z");
//double background_amu = cfg.lookup("backgroundPlasmaProfiles.amu");
double *background_flow;
double *maxDensity;
double *maxTemp_eV;

#ifdef __HIPCC__
    cout<<"Using THRUST"<<endl;
#else
    cout<<"Not using THRUST"<<endl;
#endif

/*Setting& backgroundPlasma = cfg.lookup("backgroundPlasma");
int nS = backgroundPlasma["Z"].getLength();

Setting& diagnostics = cfg.lookup("diagnostics");


background_Z = new int[nS];
background_amu = new double[nS];
background_flow = new double[nS];
maxDensity = new double[nS];
maxTemp_eV = new double[nS];

for(int i=0; i<nS; i++)
{
background_Z[i] = backgroundPlasma["Z"][i];
background_amu[i] = backgroundPlasma["amu"][i];
background_flow[i] = backgroundPlasma["flow"]["fractionOfThermalVelocity"][i];
maxDensity[i] = backgroundPlasma["density"]["max"][i];
maxTemp_eV[i] = backgroundPlasma["temp"]["max"][i];
}
*/
    double x = cfg.lookup("impurityParticleSource.initialConditions.x_start");
    double y = cfg.lookup("impurityParticleSource.initialConditions.y_start");
    double z = cfg.lookup("impurityParticleSource.initialConditions.z_start");
    
    double Ex = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_x_start");
    double Ey = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_y_start");
    double Ez = cfg.lookup("impurityParticleSource.initialConditions.energy_eV_z_start");
    
    double amu = cfg.lookup("impurityParticleSource.initialConditions.impurity_amu");
    double Z = cfg.lookup("impurityParticleSource.initialConditions.impurity_Z");
    double charge = cfg.lookup("impurityParticleSource.initialConditions.charge");
    /*
    double **SurfaceBins;
    double **SurfaceBinsCharge;
    double **SurfaceBinsEnergy;
    double **SurfaceBinsErosion;
    
    SurfaceBins = new double*[nY];
    SurfaceBinsCharge = new double*[nY];
    SurfaceBinsEnergy = new double*[nY];
    SurfaceBinsErosion = new double*[nY];

    SurfaceBins[0] = new double[nY*nZ];
    SurfaceBinsCharge[0] = new double[nY*nZ];
    SurfaceBinsEnergy[0] = new double[nY*nZ];
    SurfaceBinsErosion[0] = new double[nY*nZ];
            
    for(int i=0 ; i<nY ; i++)
    {
        SurfaceBins[i] = &SurfaceBins[0][i*nZ];
        SurfaceBinsCharge[i] = &SurfaceBinsCharge[0][i*nZ];
        SurfaceBinsEnergy[i] = &SurfaceBinsEnergy[0][i*nZ];
        SurfaceBinsErosion[i] = &SurfaceBinsErosion[0][i*nZ];               
        for(int j=0 ; j<nZ ; j++)
        {
            SurfaceBins[i][j] = 0;
            SurfaceBinsCharge[i][j] = 0;
            SurfaceBinsEnergy[i][j] = 0;
            SurfaceBinsErosion[i][j] = 0;
        }
    }
*/    
    double dt;
    double nPtsPerGyroOrbit = cfg.lookup("timeStep.nPtsPerGyroOrbit");
    dt = 1e-6/nPtsPerGyroOrbit;

    int nP = cfg.lookup("impurityParticleSource.nP");
    cout << "Number of particles: " << nP << endl;              
    long nParticles = nP;
    int nT = cfg.lookup("timeStep.nT");
    cout << "Number of time steps: " << nT << " With dt = " << dt << endl; 
    
//    int surfaceIndexY;
//    int surfaceIndexZ;
#if PARTICLE_SOURCE == 0
    Particle p1(x,y,z,Ex,Ey,Ez,Z,amu,charge);
#ifdef __HIPCC__
      thrust::host_vector<Particle> hostCudaParticleVector(nParticles,p1);
#else
        std::vector<Particle> hostCudaParticleVector(nParticles,p1);
#endif
#elif PARTICLE_SOURCE == 1
    double impurity_Z = cfg.lookup("impurityParticleSource.Z");
    int nImpurityBoundaries = 0;
    for (int i=0; i<nLines;i++)
    {
        if(hostBoundaryVector[i].Z == impurity_Z)
        {
            nImpurityBoundaries++;
        }
    }
    std::cout << "n Impurity Boundaries to launch from " << nImpurityBoundaries << std::endl;
    std::vector<int> boundaryIndex_ImpurityLaunch(nImpurityBoundaries);

    int count = 0;
    for (int i=0; i<nLines;i++)
    {
        if(hostBoundaryVector[i].Z == impurity_Z)
        {
            boundaryIndex_ImpurityLaunch[count] = i;
            count++;
            std::cout << "Boundary indices " << i << std::endl;
        }
    }
    
    int impuritiesPerBoundary = nP/nImpurityBoundaries;
#ifdef __HIPCC__
      thrust::host_vector<Particle> hostCudaParticleVector(nParticles);
#else
        std::vector<Particle> hostCudaParticleVector(nParticles);
#endif
    std::uniform_real_distribution<float> distributionForSeeds(0,1e6);
#if FIXEDSEEDS ==0
    std::random_device randDevice;
    std::default_random_engine generator0(randDevice());
#else
    double randDevice = 6.5298E+5;
    std::default_random_engine generator0(randDevice);
#endif
    
    std::vector<float> boundarySeeds0(4*nImpurityBoundaries);
    std::generate( boundarySeeds0.begin(), boundarySeeds0.end(), [&]() { return distributionForSeeds(generator0); } );
    std::uniform_real_distribution<double> dist01(0.0, 1.0);
    double rand0 = 0.0;
    double rand1 = 0.0;
    double rand2 = 0.0;
    double rand3 = 0.0;

    std::vector<std::mt19937> s0(4*nImpurityBoundaries);
    
    double E0 = 0.0;
//Create Thompson Distribution
    double surfaceBindingEnergy = cfg.lookup("impurityParticleSource.source_material_SurfaceBindingEnergy");
    std::cout << "surface binding energy " << surfaceBindingEnergy << std::endl;
    int nThompDistPoints = 200;
    double max_Energy = 100.0;
    std::vector<double> ThompsonDist(nThompDistPoints),CumulativeDFThompson(nThompDistPoints);
    for(int i=0;i<nThompDistPoints;i++)
        {
            ThompsonDist[i] = (i*max_Energy/nThompDistPoints)/pow((i*max_Energy/nThompDistPoints) + surfaceBindingEnergy,3);
            if(i==0)
            {
                CumulativeDFThompson[i] = ThompsonDist[i]; 
            }
            else
            {
                CumulativeDFThompson[i] = CumulativeDFThompson[i-1]+ThompsonDist[i];
            }
        }
    for(int i=0;i<nThompDistPoints;i++)
        {
            CumulativeDFThompson[i] = CumulativeDFThompson[i]/CumulativeDFThompson[nThompDistPoints-1];
            //std::cout << "energy and CDF" << i*max_Energy/nThompDistPoints << " " << CumulativeDFThompson[i] << std::endl;
        }

    for(int j=0; j<4*nImpurityBoundaries;j++)
        {
            std::mt19937  s(boundarySeeds0[j]);
            s0[j] = s;
        }
    // Particle p1(0.0,0.0,0.0,0.0,0.0,0.0,0,0.0);
    for (int i=0; i< nImpurityBoundaries;i++)
    {
        for(int j=0; j<impuritiesPerBoundary; j++)
        {
            //Set boundary interval, properties, and random number gen
        if (i==0)
        {
            rand0 = dist01(s0[0]);
            x = hostBoundaryVector[boundaryIndex_ImpurityLaunch[i]].x1 + hostBoundaryVector[boundaryIndex_ImpurityLaunch[i]].length*rand0;//1.4290;
            //std::cout << "start pos 1 " << x << std::endl;
            z = -1.2540+0.0001;
            rand1 = dist01(s0[1]);
            rand2 = dist01(s0[2]);
            rand3 = dist01(s0[3]);
            E0 = interp1dUnstructured(rand2,nThompDistPoints, max_Energy, &CumulativeDFThompson.front());
            Ex = E0*cos(3.1415*rand1)*sin(3.1415*rand3);
            Ey = E0*cos(3.1415*rand3);
            Ez = E0*sin(3.1415*rand1)*sin(3.1415*rand3);
        }
        else
        {
            rand0 = dist01(s0[4]);
            x = hostBoundaryVector[boundaryIndex_ImpurityLaunch[i]].x1 + hostBoundaryVector[boundaryIndex_ImpurityLaunch[i]].length*rand0;
            //x = 1.3450;
            //std::cout << "start pos 2 " << x << std::endl;
            z = -1.3660+0.0001;
            rand1 = dist01(s0[5]);
            rand2 = dist01(s0[6]);
            rand3 = dist01(s0[7]);
            E0 = interp1dUnstructured(rand2,nThompDistPoints, max_Energy, &CumulativeDFThompson.front());
            Ex = E0*cos(3.1415*rand1)*sin(3.1415*rand3);
            Ey = E0*cos(3.1415*rand3);
            Ez = E0*sin(3.1415*rand1)*sin(3.1415*rand3);
        }
            
        Particle p1(x,0.0,z,Ex,0.0,Ez,74,184.0,charge);
            hostCudaParticleVector[i*impuritiesPerBoundary + j] = p1;
            //std::cout << " E0 " << E0 << std::endl;
            //std::cout << "vy " << hostCudaParticleVector[i*impuritiesPerBoundary + j].vy << " " << Ey << std::endl;
            //std::cout << "vx " << hostCudaParticleVector[i*impuritiesPerBoundary + j].vx << " " << Ex << std::endl;
            //std::cout << "vz " << hostCudaParticleVector[i*impuritiesPerBoundary + j].vz << " " << Ez << std::endl;
        }
    }
#endif


#if GEOM_TRACE > 0       
            std::uniform_real_distribution<float> dist2(0,1);
            std::random_device rd2;
            std::cout << "Randomizing velocities to trace geometry. " << std::endl;
       
      for (int i=0 ; i<nParticles ; i++)
            {   double theta = dist2(rd2)*2*3.1415;
                double phi = dist2(rd2)*3.1415;
                double mag = 2e3;
                hostCudaParticleVector[i].vx = mag*cos(theta)*sin(phi);
                hostCudaParticleVector[i].vy = mag*sin(theta)*sin(phi);
                hostCudaParticleVector[i].vz = mag*cos(phi);
            }
#endif
#if PARTICLE_TRACKS > 0
int subSampleFac = 10;
double **positionHistoryX;
double **positionHistoryY;
double **positionHistoryZ;
double **velocityHistoryX;
double **velocityHistoryY;
double **velocityHistoryZ;
positionHistoryX = new double* [nP];
positionHistoryY = new double* [nP];
positionHistoryZ = new double* [nP];
velocityHistoryX = new double* [nP];
velocityHistoryY = new double* [nP];
velocityHistoryZ = new double* [nP];
positionHistoryX[0] = new double [nT*nP/subSampleFac];
positionHistoryY[0] = new double [nT*nP/subSampleFac];
positionHistoryZ[0] = new double [nT*nP/subSampleFac];
velocityHistoryX[0] = new double [nT*nP/subSampleFac];
velocityHistoryY[0] = new double [nT*nP/subSampleFac];
velocityHistoryZ[0] = new double [nT*nP/subSampleFac];
    for(int i=0 ; i<nP ; i++)
    {
        positionHistoryX[i] = &positionHistoryX[0][i*nT/subSampleFac];
        positionHistoryY[i] = &positionHistoryY[0][i*nT/subSampleFac];
        positionHistoryZ[i] = &positionHistoryZ[0][i*nT/subSampleFac];
        velocityHistoryX[i] = &velocityHistoryX[0][i*nT/subSampleFac];
        velocityHistoryY[i] = &velocityHistoryY[0][i*nT/subSampleFac];
        velocityHistoryZ[i] = &velocityHistoryZ[0][i*nT/subSampleFac];
        for(int j=0 ; j<nT/subSampleFac ; j++)
        {
            positionHistoryX[i][j] = 0;
            positionHistoryY[i][j] = 0;
            positionHistoryZ[i][j] = 0;
            velocityHistoryX[i][j] = 0;
            velocityHistoryY[i][j] = 0;
            velocityHistoryZ[i][j] = 0;
        }
    }
#endif   
            cpu_timer timer;

#ifdef __HIPCC__
    thrust::device_vector<Particle> deviceCudaParticleVector = hostCudaParticleVector;
#endif

    std::uniform_real_distribution<float> dist(0,1e6);
#if FIXEDSEEDS == 0
        std::random_device rd;
        std::default_random_engine generator(rd());
#endif

#if USEIONIZATION > 0
#if FIXEDSEEDS ==1
    double ionization_seeds = cfg.lookup("operators.ionization.seed");
        std::default_random_engine generator(ionization_seeds);
#endif
    std::vector<float> seeds0(nP);
    std::generate( seeds0.begin(), seeds0.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
    thrust::device_vector<float> deviceSeeds0 = seeds0;
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds0.begin(), deviceCudaParticleVector.begin(), randInit(0) );
#else
    std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds0.begin(), hostCudaParticleVector.begin(), randInit(0) );
#endif
#endif

#if USERECOMBINATION > 0
        std::vector<float> seeds1(nP);
        std::generate( seeds1.begin(), seeds1.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds1 = seeds1;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds1.begin(), deviceCudaParticleVector.begin(), randInit(1) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds1.begin(), hostCudaParticleVector.begin(), randInit(1) );
#endif
#endif

#if USEPERPDIFFUSION > 0
        std::vector<float> seeds2(nP);
        std::generate( seeds2.begin(), seeds2.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds2 = seeds2;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds2.begin(), deviceCudaParticleVector.begin(), randInit(2) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds2.begin(), hostCudaParticleVector.begin(), randInit(2) );
#endif
#endif

#if USECOULOMBCOLLISIONS > 0
        std::vector<float> seeds3(nP),seeds4(nP),seeds5(nP);
        std::generate( seeds3.begin(), seeds3.end(), [&]() { return dist(generator); } );
    std::generate( seeds4.begin(), seeds4.end(), [&]() { return dist(generator); } );
    std::generate( seeds5.begin(), seeds5.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds3 = seeds3,deviceSeeds4 = seeds4,deviceSeeds5 = seeds5;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds3.begin(), deviceCudaParticleVector.begin(), randInit(3) );
    thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds4.begin(), deviceCudaParticleVector.begin(), randInit(4) );
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds5.begin(), deviceCudaParticleVector.begin(), randInit(5) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds3.begin(), hostCudaParticleVector.begin(), randInit(3) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds4.begin(), hostCudaParticleVector.begin(), randInit(4) );
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds5.begin(), hostCudaParticleVector.begin(), randInit(5) );
#endif
#endif

#if USESURFACEMODEL > 0
        std::vector<float> seeds6(nP);
        std::generate( seeds6.begin(), seeds6.end(), [&]() { return dist(generator); } );
#ifdef __HIPCC__
        thrust::device_vector<float> deviceSeeds6 = seeds6;
        thrust::transform(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(),
                    deviceSeeds6.begin(), deviceCudaParticleVector.begin(), randInit(6) );
#else
        std::transform(hostCudaParticleVector.begin(), hostCudaParticleVector.end(),
                    seeds6.begin(), hostCudaParticleVector.begin(), randInit(6) );
#endif
#endif
    double moveTime = 0.0;
    double geomCheckTime = 0.0;
    double ionizTime = 0.0;
    cpu_times copyToDeviceTime = timer.elapsed();
    std::cout << "Initialize rand state and copyToDeviceTime: " << copyToDeviceTime.wall*1e-9 << '\n';
    for(int tt=0; tt< nT; tt++)
    {
#ifdef __HIPCC__
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), move_boris(dt,BoundaryDevicePointer, nLines,nR_Bfield,nZ_Bfield, BfieldGridRDevicePointer,BfieldGridZDevicePointer,
    BfieldRDevicePointer,BfieldZDevicePointer,BfieldTDevicePointer));
        try {
            thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), geometry_check(nLines,BoundaryDevicePointer) );
        }
        catch (thrust::system_error &e) {
            std::cerr << "Thrust system error: " << e.what() << std::endl;
            exit(-1);
        }
#if USEIONIZATION > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), ionize(dt) );
#endif
#if USERECOMBINATION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), crossFieldDiffusion(dt,perpDiffusionCoeff));
#endif
#if USECOULOMBCOLLISIONS > 0
    thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), coulombCollisions(dt) );
#endif
#if USETHERMALFORCE > 0
        thrust::for_each(deviceCudaParticleVector.begin(), deviceCudaParticleVector.end(), thermalForce(dt) );
#endif
#else
cpu_times moveTime0 = timer.elapsed();
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), move_boris(dt,hostBoundaryVector,nLines, nR_Bfield,nZ_Bfield, &bfieldGridr.front(),&bfieldGridz.front(),
                &br.front(),&bz.front(),&bt.front()));
cpu_times moveTime1 = timer.elapsed();
moveTime = moveTime + (moveTime1.wall - moveTime0.wall);

cpu_times geomTime0 = timer.elapsed();
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), geometry_check(nLines,hostBoundaryVector) );
cpu_times geomTime1 = timer.elapsed();
geomCheckTime = geomCheckTime + (geomTime1.wall - geomTime0.wall);
#if USEIONIZATION > 0
cpu_times ionizTime0 = timer.elapsed();
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), ionize(dt,
                nR_Dens,nZ_Dens,&DensGridr.front(),&DensGridz.front(),&ne.front(),
                nR_Temp,nZ_Temp,&TempGridr.front(),&TempGridz.front(),&te.front(),
                nTemperaturesIonize, nDensitiesIonize, &gridTemperature_Ionization.front(),
               &gridDensity_Ionization.front(), &rateCoeff_Ionization.front() ) );
cpu_times ionizTime1 = timer.elapsed();
ionizTime = ionizTime + (ionizTime1.wall - ionizTime0.wall);
#endif
#if USERECOMBINATION > 0
    std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), recombine(dt) );
#endif
#if USEPERPDIFFUSION > 0
    //std::cout<< "Perp diffusion loop " << perpDiffusionCoeff << std::endl;
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), crossFieldDiffusion(dt,perpDiffusionCoeff,nR_Bfield,nZ_Bfield, &bfieldGridr.front(),&bfieldGridz.front(),
                                    &br.front(),&bz.front(),&bt.front()));
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), geometry_check(nLines,hostBoundaryVector) );
#endif
#if USECOULOMBCOLLISIONS > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), coulombCollisions(dt,nR_flowV,nZ_flowV,&flowVGridr.front(),&flowVGridz.front(),&flowVr.front(),&flowVz.front(),
                    &flowVt.front(),
                nR_Dens,nZ_Dens,&DensGridr.front(),&DensGridz.front(),&ne.front(),
                nR_Temp,nZ_Temp,&TempGridr.front(),&TempGridz.front(),&te.front(),
                background_Z,background_amu,nR_Bfield,nZ_Bfield, &bfieldGridr.front(),
                &bfieldGridz.front(),&br.front(),&bz.front(),&bt.front()));
#endif
#if USETHERMALFORCE > 0
        std::for_each(hostCudaParticleVector.begin(), hostCudaParticleVector.end(), thermalForce(dt,background_amu,nR_gradT,nZ_gradT,&gradTGridr.front(),&gradTGridz.front(),
                    &gradTiR.front(),&gradTiZ.front(),&gradTeR.front(),&gradTeZ.front() ) );
#endif
#if PARTICLE_TRACKS >0
if (tt % subSampleFac == 0)  
{    
        for(int i=0;i<nP;i++)
        {
            positionHistoryX[i][tt/subSampleFac] = hostCudaParticleVector[i].xprevious;
            positionHistoryY[i][tt/subSampleFac] = hostCudaParticleVector[i].yprevious;
            positionHistoryZ[i][tt/subSampleFac] = hostCudaParticleVector[i].zprevious;
            velocityHistoryX[i][tt/subSampleFac] = hostCudaParticleVector[i].vx;
            velocityHistoryY[i][tt/subSampleFac] = hostCudaParticleVector[i].vy;
            velocityHistoryZ[i][tt/subSampleFac] = hostCudaParticleVector[i].vz;
        }
}
#endif
#endif
    }
    cpu_times ionizeTimeGPU = timer.elapsed();
    std::cout << "Particle Moving Time: " << ionizeTimeGPU.wall*1e-9 << '\n';

#ifdef __HIPCC__
    hostCudaParticleVector = deviceCudaParticleVector;
#endif

    for(int i=0; i < hostCudaParticleVector.size(); i++){
       //std::cout << " final pos" <<  i << " " <<hostCudaParticleVector[i].x << " " << hostCudaParticleVector[i].y << " " << hostCudaParticleVector[i].z << std::endl;
        /*if(hostCudaParticleVector[i].hitWall == 1){
        surfaceIndexY = int(floor((hostCudaParticleVector[i].y - yMin)/(yMax - yMin)*(nY) + 0.0f));
        surfaceIndexZ = int(floor((hostCudaParticleVector[i].z - zMin)/(zMax - zMin)*(nZ) + 0.0f));
        SurfaceBins[surfaceIndexY][surfaceIndexZ] +=  1.0 ;

        SurfaceBinsCharge[surfaceIndexY][surfaceIndexZ] += hostCudaParticleVector[i].Z ;
        SurfaceBinsEnergy[surfaceIndexY][surfaceIndexZ] += 0.5*hostCudaParticleVector[i].amu*1.6737236e-27*(hostCudaParticleVector[i].vx*hostCudaParticleVector[i].vx +  hostCudaParticleVector[i].vy*hostCudaParticleVector[i].vy+ hostCudaParticleVector[i].vz*hostCudaParticleVector[i].vz)/1.60217662e-19;
        } */ 
    }

//    OUTPUT( outname,nY, nZ, SurfaceBins);
//    OUTPUT( outnameCharge,nY, nZ, SurfaceBinsCharge);
//    OUTPUT( outnameEnergy,nY, nZ, SurfaceBinsEnergy);

    ofstream outfile2;
    outfile2.open ("positions.m");
    for(int i=1 ; i<=nP ; i++)
      {
        outfile2 << "Pos( " << i<< ",:) = [ " ;
        outfile2 << hostCudaParticleVector[i-1].x << " " << hostCudaParticleVector[i-1].y << " " << hostCudaParticleVector[i-1].z << " ];" << std::endl;
      }
       outfile2.close();
#if PARTICLE_TRACKS > 0
/*char outnameX[] = "positionHistoryX.m";
OUTPUT( outnameX,nP, nT/subSampleFac, positionHistoryX);
char outnameY[] = "positionHistoryY.m";
OUTPUT( outnameY,nP, nT/subSampleFac, positionHistoryY);
char outnameZ[] = "positionHistoryZ.m";
OUTPUT( outnameZ,nP, nT/subSampleFac, positionHistoryZ);
char outnameVX[] = "velocityHistoryX.m";
OUTPUT( outnameVX,nP, nT/subSampleFac,velocityHistoryX);
char outnameVY[] = "velocityHistoryY.m";
OUTPUT( outnameVY,nP, nT/subSampleFac, velocityHistoryY);
char outnameVZ[] = "velocityHistoryZ.m";
OUTPUT( outnameVZ,nP, nT/subSampleFac, velocityHistoryZ);
*/
// Write netCDF output for histories

NcFile ncFile("history.nc", NcFile::replace);
NcDim nc_nT = ncFile.addDim("nT",nT/subSampleFac);
NcDim nc_nP = ncFile.addDim("nP",nP);
vector<NcDim> dims;
dims.push_back(nc_nP);
dims.push_back(nc_nT);

NcVar nc_x = ncFile.addVar("x",ncDouble,dims);
NcVar nc_y = ncFile.addVar("y",ncDouble,dims);
NcVar nc_z = ncFile.addVar("z",ncDouble,dims);

NcVar nc_vx = ncFile.addVar("vx",ncDouble,dims);
NcVar nc_vy = ncFile.addVar("vy",ncDouble,dims);
NcVar nc_vz = ncFile.addVar("vz",ncDouble,dims);

nc_x.putVar(positionHistoryX[0]);
nc_y.putVar(positionHistoryY[0]);
nc_z.putVar(positionHistoryZ[0]);

nc_vx.putVar(velocityHistoryX[0]);
nc_vy.putVar(velocityHistoryY[0]);
nc_vz.putVar(velocityHistoryZ[0]);

#endif

#ifdef __HIPCC__
    hipDeviceSynchronize();
#endif

    cpu_times copyToHostTime = timer.elapsed();

    cpu_times createParticlesTimeCPU = timer.elapsed();
    std::cout << "Copy to host, bin and output time: " << (createParticlesTimeCPU.wall-copyToHostTime.wall)*1e-9 << '\n';
    std::cout << "Total ODE integration time: " << moveTime*1e-9 << '\n';
    std::cout << "Total geometry checking time: " << geomCheckTime*1e-9 << '\n';
    std::cout << "Total ionization time: " << ionizTime*1e-9 << '\n';
    return 0;
}
